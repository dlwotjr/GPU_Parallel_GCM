#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include<stdint.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <string.h>
__host__ __device__ static void print_hex(const uint8_t* data, int len)
{
    for (int i = 0; i < len; i++) {
        printf("%02X", data[i]);
        if (i % 16 == 15)printf("\n");
    }
    printf("\n");
}
typedef struct lea_key_st
{
    unsigned int rk[192];
    unsigned int round;
} LEA_KEY;
typedef struct lea_gcm_ctx
{
    //uint8_t sub_h[4][16];
    uint8_t h[256][16];
    //uint8_t sub_GHASH[256];
    uint8_t ctr[16];
    uint8_t ek0[16];
    uint8_t tbl[16];   /* tag block */
    uint8_t yn[16];   /* last encrypted block */
    LEA_KEY key;

    int yn_used, aad_len, ct_len;
    int ghash_block;
    int is_encrypt;
} LEA_GCM_CTX;
__host__ __device__ unsigned int ROL(unsigned int u, int k) {
    u = (u << k) | (u >> (32 - k));
    return u;
}
__host__ __device__  unsigned int ROR(unsigned int u, int k) {
    u = (u >> k) | (u << (32 - k));
    return u;
}
__constant__  const unsigned int delta[8][36] = {
   { 0xc3efe9db, 0x87dfd3b7, 0x0fbfa76f, 0x1f7f4ede, 0x3efe9dbc, 0x7dfd3b78, 0xfbfa76f0, 0xf7f4ede1,
   0xefe9dbc3, 0xdfd3b787, 0xbfa76f0f, 0x7f4ede1f, 0xfe9dbc3e, 0xfd3b787d, 0xfa76f0fb, 0xf4ede1f7,
   0xe9dbc3ef, 0xd3b787df, 0xa76f0fbf, 0x4ede1f7f, 0x9dbc3efe, 0x3b787dfd, 0x76f0fbfa, 0xede1f7f4,
   0xdbc3efe9, 0xb787dfd3, 0x6f0fbfa7, 0xde1f7f4e, 0xbc3efe9d, 0x787dfd3b, 0xf0fbfa76, 0xe1f7f4eD,
   0xc3efe9db, 0x87dfd3b7, 0x0fbfa76f, 0x1f7f4ede },
   { 0x44626b02, 0x88c4d604, 0x1189ac09, 0x23135812, 0x4626b024, 0x8c4d6048, 0x189ac091, 0x31358122,
   0x626b0244, 0xc4d60488, 0x89ac0911, 0x13581223, 0x26b02446, 0x4d60488c, 0x9ac09118, 0x35812231,
   0x6b024462, 0xd60488c4, 0xac091189, 0x58122313, 0xb0244626, 0x60488c4d, 0xc091189a, 0x81223135,
   0x0244626b, 0x0488c4d6, 0x091189ac, 0x12231358, 0x244626b0, 0x488c4d60, 0x91189ac0, 0x22313581,
   0x44626b02, 0x88c4d604, 0x1189ac09, 0x23135812 },
   { 0x79e27c8a, 0xf3c4f914, 0xe789f229, 0xcf13e453, 0x9e27c8a7, 0x3c4f914f, 0x789f229e, 0xf13e453c,
   0xe27c8a79, 0xc4f914f3, 0x89f229e7, 0x13e453cf, 0x27c8a79e, 0x4f914f3c, 0x9f229e78, 0x3e453cf1,
   0x7c8a79e2, 0xf914f3c4, 0xf229e789, 0xe453cf13, 0xc8a79e27, 0x914f3c4f, 0x229e789f, 0x453cf13e,
   0x8a79e27c, 0x14f3c4f9, 0x29e789f2, 0x53cf13e4, 0xa79e27c8, 0x4f3c4f91, 0x9e789f22, 0x3cf13e45,
   0x79e27c8a, 0xf3c4f914, 0xe789f229, 0xcf13e453 },
   { 0x78df30ec, 0xf1be61d8, 0xe37cc3b1, 0xc6f98763, 0x8df30ec7, 0x1be61d8f, 0x37cc3b1e, 0x6f98763c,
   0xdf30ec78, 0xbe61d8f1, 0x7cc3b1e3, 0xf98763c6, 0xf30ec78d, 0xe61d8f1b, 0xcc3b1e37, 0x98763c6f,
   0x30ec78df, 0x61d8f1be, 0xc3b1e37c, 0x8763c6f9, 0x0ec78df3, 0x1d8f1be6, 0x3b1e37cc, 0x763c6f98,
   0xec78df30, 0xd8f1be61, 0xb1e37cc3, 0x63c6f987, 0xc78df30e, 0x8f1be61d, 0x1e37cc3b, 0x3c6f9876,
   0x78df30ec, 0xf1be61d8, 0xe37cc3b1, 0xc6f98763 },
   { 0x715ea49e, 0xe2bd493c, 0xc57a9279, 0x8af524f3, 0x15ea49e7, 0x2bd493ce, 0x57a9279c, 0xaf524f38,
   0x5ea49e71, 0xbd493ce2, 0x7a9279c5, 0xf524f38a, 0xea49e715, 0xd493ce2b, 0xa9279c57, 0x524f38af,
   0xa49e715e, 0x493ce2bd, 0x9279c57a, 0x24f38af5, 0x49e715ea, 0x93ce2bd4, 0x279c57a9, 0x4f38af52,
   0x9e715ea4, 0x3ce2bd49, 0x79c57a92, 0xf38af524, 0xe715ea49, 0xce2bd493, 0x9c57a927, 0x38af524f,
   0x715ea49e, 0xe2bd493c, 0xc57a9279, 0x8af524f3 },
   { 0xc785da0a, 0x8f0bb415, 0x1e17682b, 0x3c2ed056, 0x785da0ac, 0xf0bb4158, 0xe17682b1, 0xc2ed0563,
   0x85da0ac7, 0x0bb4158f, 0x17682b1e, 0x2ed0563c, 0x5da0ac78, 0xbb4158f0, 0x7682b1e1, 0xed0563c2,
   0xda0ac785, 0xb4158f0b, 0x682b1e17, 0xd0563c2e, 0xa0ac785d, 0x4158f0bb, 0x82b1e176, 0x0563c2ed,
   0x0ac785da, 0x158f0bb4, 0x2b1e1768, 0x563c2ed0, 0xac785da0, 0x58f0bb41, 0xb1e17682, 0x63c2ed05,
   0xc785da0a, 0x8f0bb415, 0x1e17682b, 0x3c2ed056 },
   { 0xe04ef22a, 0xc09de455, 0x813bc8ab, 0x02779157, 0x04ef22ae, 0x09de455c, 0x13bc8ab8, 0x27791570,
   0x4ef22ae0, 0x9de455c0, 0x3bc8ab81, 0x77915702, 0xef22ae04, 0xde455c09, 0xbc8ab813, 0x79157027,
   0xf22ae04e, 0xe455c09d, 0xc8ab813b, 0x91570277, 0x22ae04ef, 0x455c09de, 0x8ab813bc, 0x15702779,
   0x2ae04ef2, 0x55c09de4, 0xab813bc8, 0x57027791, 0xae04ef22, 0x5c09de45, 0xb813bc8a, 0x70277915,
   0xe04ef22a, 0xc09de455, 0x813bc8ab, 0x02779157 },
   { 0xe5c40957, 0xcb8812af, 0x9710255f, 0x2e204abf, 0x5c40957e, 0xb8812afc, 0x710255f9, 0xe204abf2,
   0xc40957e5, 0x8812afcb, 0x10255f97, 0x204abf2e, 0x40957e5c, 0x812afcb8, 0x0255f971, 0x04abf2e2,
   0x0957e5c4, 0x12afcb88, 0x255f9710, 0x4abf2e20, 0x957e5c40, 0x2afcb881, 0x55f97102, 0xabf2e204,
   0x57e5c409, 0xafcb8812, 0x5f971025, 0xbf2e204a, 0x7e5c4095, 0xfcb8812a, 0xf9710255, 0xf2e204ab,
   0xe5c40957, 0xcb8812af, 0x9710255f, 0x2e204abf }
};
__host__ __device__  void lea_set_key(LEA_KEY* key, const uint8_t* mk, unsigned int mk_len)
{
    if (!key)
        return;
    else if (!mk)
        return;

    switch (mk_len)
    {
    case 16:
        key->rk[0] = ROL(*((unsigned int*)mk) + delta[0][0], 1);
        key->rk[6] = ROL(key->rk[0] + delta[1][1], 1);
        key->rk[12] = ROL(key->rk[6] + delta[2][2], 1);
        key->rk[18] = ROL(key->rk[12] + delta[3][3], 1);
        key->rk[24] = ROL(key->rk[18] + delta[0][4], 1);
        key->rk[30] = ROL(key->rk[24] + delta[1][5], 1);
        key->rk[36] = ROL(key->rk[30] + delta[2][6], 1);
        key->rk[42] = ROL(key->rk[36] + delta[3][7], 1);
        key->rk[48] = ROL(key->rk[42] + delta[0][8], 1);
        key->rk[54] = ROL(key->rk[48] + delta[1][9], 1);
        key->rk[60] = ROL(key->rk[54] + delta[2][10], 1);
        key->rk[66] = ROL(key->rk[60] + delta[3][11], 1);
        key->rk[72] = ROL(key->rk[66] + delta[0][12], 1);
        key->rk[78] = ROL(key->rk[72] + delta[1][13], 1);
        key->rk[84] = ROL(key->rk[78] + delta[2][14], 1);
        key->rk[90] = ROL(key->rk[84] + delta[3][15], 1);
        key->rk[96] = ROL(key->rk[90] + delta[0][16], 1);
        key->rk[102] = ROL(key->rk[96] + delta[1][17], 1);
        key->rk[108] = ROL(key->rk[102] + delta[2][18], 1);
        key->rk[114] = ROL(key->rk[108] + delta[3][19], 1);
        key->rk[120] = ROL(key->rk[114] + delta[0][20], 1);
        key->rk[126] = ROL(key->rk[120] + delta[1][21], 1);
        key->rk[132] = ROL(key->rk[126] + delta[2][22], 1);
        key->rk[138] = ROL(key->rk[132] + delta[3][23], 1);

        key->rk[1] = key->rk[3] = key->rk[5] = ROL(*((unsigned int*)mk + 1) + delta[0][1], 3);
        key->rk[7] = key->rk[9] = key->rk[11] = ROL(key->rk[1] + delta[1][2], 3);
        key->rk[13] = key->rk[15] = key->rk[17] = ROL(key->rk[7] + delta[2][3], 3);
        key->rk[19] = key->rk[21] = key->rk[23] = ROL(key->rk[13] + delta[3][4], 3);
        key->rk[25] = key->rk[27] = key->rk[29] = ROL(key->rk[19] + delta[0][5], 3);
        key->rk[31] = key->rk[33] = key->rk[35] = ROL(key->rk[25] + delta[1][6], 3);
        key->rk[37] = key->rk[39] = key->rk[41] = ROL(key->rk[31] + delta[2][7], 3);
        key->rk[43] = key->rk[45] = key->rk[47] = ROL(key->rk[37] + delta[3][8], 3);
        key->rk[49] = key->rk[51] = key->rk[53] = ROL(key->rk[43] + delta[0][9], 3);
        key->rk[55] = key->rk[57] = key->rk[59] = ROL(key->rk[49] + delta[1][10], 3);
        key->rk[61] = key->rk[63] = key->rk[65] = ROL(key->rk[55] + delta[2][11], 3);
        key->rk[67] = key->rk[69] = key->rk[71] = ROL(key->rk[61] + delta[3][12], 3);
        key->rk[73] = key->rk[75] = key->rk[77] = ROL(key->rk[67] + delta[0][13], 3);
        key->rk[79] = key->rk[81] = key->rk[83] = ROL(key->rk[73] + delta[1][14], 3);
        key->rk[85] = key->rk[87] = key->rk[89] = ROL(key->rk[79] + delta[2][15], 3);
        key->rk[91] = key->rk[93] = key->rk[95] = ROL(key->rk[85] + delta[3][16], 3);
        key->rk[97] = key->rk[99] = key->rk[101] = ROL(key->rk[91] + delta[0][17], 3);
        key->rk[103] = key->rk[105] = key->rk[107] = ROL(key->rk[97] + delta[1][18], 3);
        key->rk[109] = key->rk[111] = key->rk[113] = ROL(key->rk[103] + delta[2][19], 3);
        key->rk[115] = key->rk[117] = key->rk[119] = ROL(key->rk[109] + delta[3][20], 3);
        key->rk[121] = key->rk[123] = key->rk[125] = ROL(key->rk[115] + delta[0][21], 3);
        key->rk[127] = key->rk[129] = key->rk[131] = ROL(key->rk[121] + delta[1][22], 3);
        key->rk[133] = key->rk[135] = key->rk[137] = ROL(key->rk[127] + delta[2][23], 3);
        key->rk[139] = key->rk[141] = key->rk[143] = ROL(key->rk[133] + delta[3][24], 3);

        key->rk[2] = ROL(*((unsigned int*)mk + 2) + delta[0][2], 6);
        key->rk[8] = ROL(key->rk[2] + delta[1][3], 6);
        key->rk[14] = ROL(key->rk[8] + delta[2][4], 6);
        key->rk[20] = ROL(key->rk[14] + delta[3][5], 6);
        key->rk[26] = ROL(key->rk[20] + delta[0][6], 6);
        key->rk[32] = ROL(key->rk[26] + delta[1][7], 6);
        key->rk[38] = ROL(key->rk[32] + delta[2][8], 6);
        key->rk[44] = ROL(key->rk[38] + delta[3][9], 6);
        key->rk[50] = ROL(key->rk[44] + delta[0][10], 6);
        key->rk[56] = ROL(key->rk[50] + delta[1][11], 6);
        key->rk[62] = ROL(key->rk[56] + delta[2][12], 6);
        key->rk[68] = ROL(key->rk[62] + delta[3][13], 6);
        key->rk[74] = ROL(key->rk[68] + delta[0][14], 6);
        key->rk[80] = ROL(key->rk[74] + delta[1][15], 6);
        key->rk[86] = ROL(key->rk[80] + delta[2][16], 6);
        key->rk[92] = ROL(key->rk[86] + delta[3][17], 6);
        key->rk[98] = ROL(key->rk[92] + delta[0][18], 6);
        key->rk[104] = ROL(key->rk[98] + delta[1][19], 6);
        key->rk[110] = ROL(key->rk[104] + delta[2][20], 6);
        key->rk[116] = ROL(key->rk[110] + delta[3][21], 6);
        key->rk[122] = ROL(key->rk[116] + delta[0][22], 6);
        key->rk[128] = ROL(key->rk[122] + delta[1][23], 6);
        key->rk[134] = ROL(key->rk[128] + delta[2][24], 6);
        key->rk[140] = ROL(key->rk[134] + delta[3][25], 6);

        key->rk[4] = ROL(*((unsigned int*)mk + 3) + delta[0][3], 11);
        key->rk[10] = ROL(key->rk[4] + delta[1][4], 11);
        key->rk[16] = ROL(key->rk[10] + delta[2][5], 11);
        key->rk[22] = ROL(key->rk[16] + delta[3][6], 11);
        key->rk[28] = ROL(key->rk[22] + delta[0][7], 11);
        key->rk[34] = ROL(key->rk[28] + delta[1][8], 11);
        key->rk[40] = ROL(key->rk[34] + delta[2][9], 11);
        key->rk[46] = ROL(key->rk[40] + delta[3][10], 11);
        key->rk[52] = ROL(key->rk[46] + delta[0][11], 11);
        key->rk[58] = ROL(key->rk[52] + delta[1][12], 11);
        key->rk[64] = ROL(key->rk[58] + delta[2][13], 11);
        key->rk[70] = ROL(key->rk[64] + delta[3][14], 11);
        key->rk[76] = ROL(key->rk[70] + delta[0][15], 11);
        key->rk[82] = ROL(key->rk[76] + delta[1][16], 11);
        key->rk[88] = ROL(key->rk[82] + delta[2][17], 11);
        key->rk[94] = ROL(key->rk[88] + delta[3][18], 11);
        key->rk[100] = ROL(key->rk[94] + delta[0][19], 11);
        key->rk[106] = ROL(key->rk[100] + delta[1][20], 11);
        key->rk[112] = ROL(key->rk[106] + delta[2][21], 11);
        key->rk[118] = ROL(key->rk[112] + delta[3][22], 11);
        key->rk[124] = ROL(key->rk[118] + delta[0][23], 11);
        key->rk[130] = ROL(key->rk[124] + delta[1][24], 11);
        key->rk[136] = ROL(key->rk[130] + delta[2][25], 11);
        key->rk[142] = ROL(key->rk[136] + delta[3][26], 11);
        break;

    case 24:
        key->rk[0] = ROL(*((unsigned int*)mk) + delta[0][0], 1);
        key->rk[6] = ROL(key->rk[0] + delta[1][1], 1);
        key->rk[12] = ROL(key->rk[6] + delta[2][2], 1);
        key->rk[18] = ROL(key->rk[12] + delta[3][3], 1);
        key->rk[24] = ROL(key->rk[18] + delta[4][4], 1);
        key->rk[30] = ROL(key->rk[24] + delta[5][5], 1);
        key->rk[36] = ROL(key->rk[30] + delta[0][6], 1);
        key->rk[42] = ROL(key->rk[36] + delta[1][7], 1);
        key->rk[48] = ROL(key->rk[42] + delta[2][8], 1);
        key->rk[54] = ROL(key->rk[48] + delta[3][9], 1);
        key->rk[60] = ROL(key->rk[54] + delta[4][10], 1);
        key->rk[66] = ROL(key->rk[60] + delta[5][11], 1);
        key->rk[72] = ROL(key->rk[66] + delta[0][12], 1);
        key->rk[78] = ROL(key->rk[72] + delta[1][13], 1);
        key->rk[84] = ROL(key->rk[78] + delta[2][14], 1);
        key->rk[90] = ROL(key->rk[84] + delta[3][15], 1);
        key->rk[96] = ROL(key->rk[90] + delta[4][16], 1);
        key->rk[102] = ROL(key->rk[96] + delta[5][17], 1);
        key->rk[108] = ROL(key->rk[102] + delta[0][18], 1);
        key->rk[114] = ROL(key->rk[108] + delta[1][19], 1);
        key->rk[120] = ROL(key->rk[114] + delta[2][20], 1);
        key->rk[126] = ROL(key->rk[120] + delta[3][21], 1);
        key->rk[132] = ROL(key->rk[126] + delta[4][22], 1);
        key->rk[138] = ROL(key->rk[132] + delta[5][23], 1);
        key->rk[144] = ROL(key->rk[138] + delta[0][24], 1);
        key->rk[150] = ROL(key->rk[144] + delta[1][25], 1);
        key->rk[156] = ROL(key->rk[150] + delta[2][26], 1);
        key->rk[162] = ROL(key->rk[156] + delta[3][27], 1);

        key->rk[1] = ROL(*((unsigned int*)mk + 1) + delta[0][1], 3);
        key->rk[7] = ROL(key->rk[1] + delta[1][2], 3);
        key->rk[13] = ROL(key->rk[7] + delta[2][3], 3);
        key->rk[19] = ROL(key->rk[13] + delta[3][4], 3);
        key->rk[25] = ROL(key->rk[19] + delta[4][5], 3);
        key->rk[31] = ROL(key->rk[25] + delta[5][6], 3);
        key->rk[37] = ROL(key->rk[31] + delta[0][7], 3);
        key->rk[43] = ROL(key->rk[37] + delta[1][8], 3);
        key->rk[49] = ROL(key->rk[43] + delta[2][9], 3);
        key->rk[55] = ROL(key->rk[49] + delta[3][10], 3);
        key->rk[61] = ROL(key->rk[55] + delta[4][11], 3);
        key->rk[67] = ROL(key->rk[61] + delta[5][12], 3);
        key->rk[73] = ROL(key->rk[67] + delta[0][13], 3);
        key->rk[79] = ROL(key->rk[73] + delta[1][14], 3);
        key->rk[85] = ROL(key->rk[79] + delta[2][15], 3);
        key->rk[91] = ROL(key->rk[85] + delta[3][16], 3);
        key->rk[97] = ROL(key->rk[91] + delta[4][17], 3);
        key->rk[103] = ROL(key->rk[97] + delta[5][18], 3);
        key->rk[109] = ROL(key->rk[103] + delta[0][19], 3);
        key->rk[115] = ROL(key->rk[109] + delta[1][20], 3);
        key->rk[121] = ROL(key->rk[115] + delta[2][21], 3);
        key->rk[127] = ROL(key->rk[121] + delta[3][22], 3);
        key->rk[133] = ROL(key->rk[127] + delta[4][23], 3);
        key->rk[139] = ROL(key->rk[133] + delta[5][24], 3);
        key->rk[145] = ROL(key->rk[139] + delta[0][25], 3);
        key->rk[151] = ROL(key->rk[145] + delta[1][26], 3);
        key->rk[157] = ROL(key->rk[151] + delta[2][27], 3);
        key->rk[163] = ROL(key->rk[157] + delta[3][28], 3);

        key->rk[2] = ROL(*((unsigned int*)mk + 2) + delta[0][2], 6);
        key->rk[8] = ROL(key->rk[2] + delta[1][3], 6);
        key->rk[14] = ROL(key->rk[8] + delta[2][4], 6);
        key->rk[20] = ROL(key->rk[14] + delta[3][5], 6);
        key->rk[26] = ROL(key->rk[20] + delta[4][6], 6);
        key->rk[32] = ROL(key->rk[26] + delta[5][7], 6);
        key->rk[38] = ROL(key->rk[32] + delta[0][8], 6);
        key->rk[44] = ROL(key->rk[38] + delta[1][9], 6);
        key->rk[50] = ROL(key->rk[44] + delta[2][10], 6);
        key->rk[56] = ROL(key->rk[50] + delta[3][11], 6);
        key->rk[62] = ROL(key->rk[56] + delta[4][12], 6);
        key->rk[68] = ROL(key->rk[62] + delta[5][13], 6);
        key->rk[74] = ROL(key->rk[68] + delta[0][14], 6);
        key->rk[80] = ROL(key->rk[74] + delta[1][15], 6);
        key->rk[86] = ROL(key->rk[80] + delta[2][16], 6);
        key->rk[92] = ROL(key->rk[86] + delta[3][17], 6);
        key->rk[98] = ROL(key->rk[92] + delta[4][18], 6);
        key->rk[104] = ROL(key->rk[98] + delta[5][19], 6);
        key->rk[110] = ROL(key->rk[104] + delta[0][20], 6);
        key->rk[116] = ROL(key->rk[110] + delta[1][21], 6);
        key->rk[122] = ROL(key->rk[116] + delta[2][22], 6);
        key->rk[128] = ROL(key->rk[122] + delta[3][23], 6);
        key->rk[134] = ROL(key->rk[128] + delta[4][24], 6);
        key->rk[140] = ROL(key->rk[134] + delta[5][25], 6);
        key->rk[146] = ROL(key->rk[140] + delta[0][26], 6);
        key->rk[152] = ROL(key->rk[146] + delta[1][27], 6);
        key->rk[158] = ROL(key->rk[152] + delta[2][28], 6);
        key->rk[164] = ROL(key->rk[158] + delta[3][29], 6);

        key->rk[3] = ROL(*((unsigned int*)mk + 3) + delta[0][3], 11);
        key->rk[9] = ROL(key->rk[3] + delta[1][4], 11);
        key->rk[15] = ROL(key->rk[9] + delta[2][5], 11);
        key->rk[21] = ROL(key->rk[15] + delta[3][6], 11);
        key->rk[27] = ROL(key->rk[21] + delta[4][7], 11);
        key->rk[33] = ROL(key->rk[27] + delta[5][8], 11);
        key->rk[39] = ROL(key->rk[33] + delta[0][9], 11);
        key->rk[45] = ROL(key->rk[39] + delta[1][10], 11);
        key->rk[51] = ROL(key->rk[45] + delta[2][11], 11);
        key->rk[57] = ROL(key->rk[51] + delta[3][12], 11);
        key->rk[63] = ROL(key->rk[57] + delta[4][13], 11);
        key->rk[69] = ROL(key->rk[63] + delta[5][14], 11);
        key->rk[75] = ROL(key->rk[69] + delta[0][15], 11);
        key->rk[81] = ROL(key->rk[75] + delta[1][16], 11);
        key->rk[87] = ROL(key->rk[81] + delta[2][17], 11);
        key->rk[93] = ROL(key->rk[87] + delta[3][18], 11);
        key->rk[99] = ROL(key->rk[93] + delta[4][19], 11);
        key->rk[105] = ROL(key->rk[99] + delta[5][20], 11);
        key->rk[111] = ROL(key->rk[105] + delta[0][21], 11);
        key->rk[117] = ROL(key->rk[111] + delta[1][22], 11);
        key->rk[123] = ROL(key->rk[117] + delta[2][23], 11);
        key->rk[129] = ROL(key->rk[123] + delta[3][24], 11);
        key->rk[135] = ROL(key->rk[129] + delta[4][25], 11);
        key->rk[141] = ROL(key->rk[135] + delta[5][26], 11);
        key->rk[147] = ROL(key->rk[141] + delta[0][27], 11);
        key->rk[153] = ROL(key->rk[147] + delta[1][28], 11);
        key->rk[159] = ROL(key->rk[153] + delta[2][29], 11);
        key->rk[165] = ROL(key->rk[159] + delta[3][30], 11);

        key->rk[4] = ROL(*((unsigned int*)mk + 4) + delta[0][4], 13);
        key->rk[10] = ROL(key->rk[4] + delta[1][5], 13);
        key->rk[16] = ROL(key->rk[10] + delta[2][6], 13);
        key->rk[22] = ROL(key->rk[16] + delta[3][7], 13);
        key->rk[28] = ROL(key->rk[22] + delta[4][8], 13);
        key->rk[34] = ROL(key->rk[28] + delta[5][9], 13);
        key->rk[40] = ROL(key->rk[34] + delta[0][10], 13);
        key->rk[46] = ROL(key->rk[40] + delta[1][11], 13);
        key->rk[52] = ROL(key->rk[46] + delta[2][12], 13);
        key->rk[58] = ROL(key->rk[52] + delta[3][13], 13);
        key->rk[64] = ROL(key->rk[58] + delta[4][14], 13);
        key->rk[70] = ROL(key->rk[64] + delta[5][15], 13);
        key->rk[76] = ROL(key->rk[70] + delta[0][16], 13);
        key->rk[82] = ROL(key->rk[76] + delta[1][17], 13);
        key->rk[88] = ROL(key->rk[82] + delta[2][18], 13);
        key->rk[94] = ROL(key->rk[88] + delta[3][19], 13);
        key->rk[100] = ROL(key->rk[94] + delta[4][20], 13);
        key->rk[106] = ROL(key->rk[100] + delta[5][21], 13);
        key->rk[112] = ROL(key->rk[106] + delta[0][22], 13);
        key->rk[118] = ROL(key->rk[112] + delta[1][23], 13);
        key->rk[124] = ROL(key->rk[118] + delta[2][24], 13);
        key->rk[130] = ROL(key->rk[124] + delta[3][25], 13);
        key->rk[136] = ROL(key->rk[130] + delta[4][26], 13);
        key->rk[142] = ROL(key->rk[136] + delta[5][27], 13);
        key->rk[148] = ROL(key->rk[142] + delta[0][28], 13);
        key->rk[154] = ROL(key->rk[148] + delta[1][29], 13);
        key->rk[160] = ROL(key->rk[154] + delta[2][30], 13);
        key->rk[166] = ROL(key->rk[160] + delta[3][31], 13);

        key->rk[5] = ROL(*((unsigned int*)mk + 5) + delta[0][5], 17);
        key->rk[11] = ROL(key->rk[5] + delta[1][6], 17);
        key->rk[17] = ROL(key->rk[11] + delta[2][7], 17);
        key->rk[23] = ROL(key->rk[17] + delta[3][8], 17);
        key->rk[29] = ROL(key->rk[23] + delta[4][9], 17);
        key->rk[35] = ROL(key->rk[29] + delta[5][10], 17);
        key->rk[41] = ROL(key->rk[35] + delta[0][11], 17);
        key->rk[47] = ROL(key->rk[41] + delta[1][12], 17);
        key->rk[53] = ROL(key->rk[47] + delta[2][13], 17);
        key->rk[59] = ROL(key->rk[53] + delta[3][14], 17);
        key->rk[65] = ROL(key->rk[59] + delta[4][15], 17);
        key->rk[71] = ROL(key->rk[65] + delta[5][16], 17);
        key->rk[77] = ROL(key->rk[71] + delta[0][17], 17);
        key->rk[83] = ROL(key->rk[77] + delta[1][18], 17);
        key->rk[89] = ROL(key->rk[83] + delta[2][19], 17);
        key->rk[95] = ROL(key->rk[89] + delta[3][20], 17);
        key->rk[101] = ROL(key->rk[95] + delta[4][21], 17);
        key->rk[107] = ROL(key->rk[101] + delta[5][22], 17);
        key->rk[113] = ROL(key->rk[107] + delta[0][23], 17);
        key->rk[119] = ROL(key->rk[113] + delta[1][24], 17);
        key->rk[125] = ROL(key->rk[119] + delta[2][25], 17);
        key->rk[131] = ROL(key->rk[125] + delta[3][26], 17);
        key->rk[137] = ROL(key->rk[131] + delta[4][27], 17);
        key->rk[143] = ROL(key->rk[137] + delta[5][28], 17);
        key->rk[149] = ROL(key->rk[143] + delta[0][29], 17);
        key->rk[155] = ROL(key->rk[149] + delta[1][30], 17);
        key->rk[161] = ROL(key->rk[155] + delta[2][31], 17);
        key->rk[167] = ROL(key->rk[161] + delta[3][0], 17);
        break;

    case 32:
        key->rk[0] = ROL(*((unsigned int*)mk) + delta[0][0], 1);
        key->rk[8] = ROL(key->rk[0] + delta[1][3], 6);
        key->rk[16] = ROL(key->rk[8] + delta[2][6], 13);
        key->rk[24] = ROL(key->rk[16] + delta[4][4], 1);
        key->rk[32] = ROL(key->rk[24] + delta[5][7], 6);
        key->rk[40] = ROL(key->rk[32] + delta[6][10], 13);
        key->rk[48] = ROL(key->rk[40] + delta[0][8], 1);
        key->rk[56] = ROL(key->rk[48] + delta[1][11], 6);
        key->rk[64] = ROL(key->rk[56] + delta[2][14], 13);
        key->rk[72] = ROL(key->rk[64] + delta[4][12], 1);
        key->rk[80] = ROL(key->rk[72] + delta[5][15], 6);
        key->rk[88] = ROL(key->rk[80] + delta[6][18], 13);
        key->rk[96] = ROL(key->rk[88] + delta[0][16], 1);
        key->rk[104] = ROL(key->rk[96] + delta[1][19], 6);
        key->rk[112] = ROL(key->rk[104] + delta[2][22], 13);
        key->rk[120] = ROL(key->rk[112] + delta[4][20], 1);
        key->rk[128] = ROL(key->rk[120] + delta[5][23], 6);
        key->rk[136] = ROL(key->rk[128] + delta[6][26], 13);
        key->rk[144] = ROL(key->rk[136] + delta[0][24], 1);
        key->rk[152] = ROL(key->rk[144] + delta[1][27], 6);
        key->rk[160] = ROL(key->rk[152] + delta[2][30], 13);
        key->rk[168] = ROL(key->rk[160] + delta[4][28], 1);
        key->rk[176] = ROL(key->rk[168] + delta[5][31], 6);
        key->rk[184] = ROL(key->rk[176] + delta[6][2], 13);

        key->rk[1] = ROL(*((unsigned int*)mk + 1) + delta[0][1], 3);
        key->rk[9] = ROL(key->rk[1] + delta[1][4], 11);
        key->rk[17] = ROL(key->rk[9] + delta[2][7], 17);
        key->rk[25] = ROL(key->rk[17] + delta[4][5], 3);
        key->rk[33] = ROL(key->rk[25] + delta[5][8], 11);
        key->rk[41] = ROL(key->rk[33] + delta[6][11], 17);
        key->rk[49] = ROL(key->rk[41] + delta[0][9], 3);
        key->rk[57] = ROL(key->rk[49] + delta[1][12], 11);
        key->rk[65] = ROL(key->rk[57] + delta[2][15], 17);
        key->rk[73] = ROL(key->rk[65] + delta[4][13], 3);
        key->rk[81] = ROL(key->rk[73] + delta[5][16], 11);
        key->rk[89] = ROL(key->rk[81] + delta[6][19], 17);
        key->rk[97] = ROL(key->rk[89] + delta[0][17], 3);
        key->rk[105] = ROL(key->rk[97] + delta[1][20], 11);
        key->rk[113] = ROL(key->rk[105] + delta[2][23], 17);
        key->rk[121] = ROL(key->rk[113] + delta[4][21], 3);
        key->rk[129] = ROL(key->rk[121] + delta[5][24], 11);
        key->rk[137] = ROL(key->rk[129] + delta[6][27], 17);
        key->rk[145] = ROL(key->rk[137] + delta[0][25], 3);
        key->rk[153] = ROL(key->rk[145] + delta[1][28], 11);
        key->rk[161] = ROL(key->rk[153] + delta[2][31], 17);
        key->rk[169] = ROL(key->rk[161] + delta[4][29], 3);
        key->rk[177] = ROL(key->rk[169] + delta[5][0], 11);
        key->rk[185] = ROL(key->rk[177] + delta[6][3], 17);

        key->rk[2] = ROL(*((unsigned int*)mk + 2) + delta[0][2], 6);
        key->rk[10] = ROL(key->rk[2] + delta[1][5], 13);
        key->rk[18] = ROL(key->rk[10] + delta[3][3], 1);
        key->rk[26] = ROL(key->rk[18] + delta[4][6], 6);
        key->rk[34] = ROL(key->rk[26] + delta[5][9], 13);
        key->rk[42] = ROL(key->rk[34] + delta[7][7], 1);
        key->rk[50] = ROL(key->rk[42] + delta[0][10], 6);
        key->rk[58] = ROL(key->rk[50] + delta[1][13], 13);
        key->rk[66] = ROL(key->rk[58] + delta[3][11], 1);
        key->rk[74] = ROL(key->rk[66] + delta[4][14], 6);
        key->rk[82] = ROL(key->rk[74] + delta[5][17], 13);
        key->rk[90] = ROL(key->rk[82] + delta[7][15], 1);
        key->rk[98] = ROL(key->rk[90] + delta[0][18], 6);
        key->rk[106] = ROL(key->rk[98] + delta[1][21], 13);
        key->rk[114] = ROL(key->rk[106] + delta[3][19], 1);
        key->rk[122] = ROL(key->rk[114] + delta[4][22], 6);
        key->rk[130] = ROL(key->rk[122] + delta[5][25], 13);
        key->rk[138] = ROL(key->rk[130] + delta[7][23], 1);
        key->rk[146] = ROL(key->rk[138] + delta[0][26], 6);
        key->rk[154] = ROL(key->rk[146] + delta[1][29], 13);
        key->rk[162] = ROL(key->rk[154] + delta[3][27], 1);
        key->rk[170] = ROL(key->rk[162] + delta[4][30], 6);
        key->rk[178] = ROL(key->rk[170] + delta[5][1], 13);
        key->rk[186] = ROL(key->rk[178] + delta[7][31], 1);

        key->rk[3] = ROL(*((unsigned int*)mk + 3) + delta[0][3], 11);
        key->rk[11] = ROL(key->rk[3] + delta[1][6], 17);
        key->rk[19] = ROL(key->rk[11] + delta[3][4], 3);
        key->rk[27] = ROL(key->rk[19] + delta[4][7], 11);
        key->rk[35] = ROL(key->rk[27] + delta[5][10], 17);
        key->rk[43] = ROL(key->rk[35] + delta[7][8], 3);
        key->rk[51] = ROL(key->rk[43] + delta[0][11], 11);
        key->rk[59] = ROL(key->rk[51] + delta[1][14], 17);
        key->rk[67] = ROL(key->rk[59] + delta[3][12], 3);
        key->rk[75] = ROL(key->rk[67] + delta[4][15], 11);
        key->rk[83] = ROL(key->rk[75] + delta[5][18], 17);
        key->rk[91] = ROL(key->rk[83] + delta[7][16], 3);
        key->rk[99] = ROL(key->rk[91] + delta[0][19], 11);
        key->rk[107] = ROL(key->rk[99] + delta[1][22], 17);
        key->rk[115] = ROL(key->rk[107] + delta[3][20], 3);
        key->rk[123] = ROL(key->rk[115] + delta[4][23], 11);
        key->rk[131] = ROL(key->rk[123] + delta[5][26], 17);
        key->rk[139] = ROL(key->rk[131] + delta[7][24], 3);
        key->rk[147] = ROL(key->rk[139] + delta[0][27], 11);
        key->rk[155] = ROL(key->rk[147] + delta[1][30], 17);
        key->rk[163] = ROL(key->rk[155] + delta[3][28], 3);
        key->rk[171] = ROL(key->rk[163] + delta[4][31], 11);
        key->rk[179] = ROL(key->rk[171] + delta[5][2], 17);
        key->rk[187] = ROL(key->rk[179] + delta[7][0], 3);

        key->rk[4] = ROL(*((unsigned int*)mk + 4) + delta[0][4], 13);
        key->rk[12] = ROL(key->rk[4] + delta[2][2], 1);
        key->rk[20] = ROL(key->rk[12] + delta[3][5], 6);
        key->rk[28] = ROL(key->rk[20] + delta[4][8], 13);
        key->rk[36] = ROL(key->rk[28] + delta[6][6], 1);
        key->rk[44] = ROL(key->rk[36] + delta[7][9], 6);
        key->rk[52] = ROL(key->rk[44] + delta[0][12], 13);
        key->rk[60] = ROL(key->rk[52] + delta[2][10], 1);
        key->rk[68] = ROL(key->rk[60] + delta[3][13], 6);
        key->rk[76] = ROL(key->rk[68] + delta[4][16], 13);
        key->rk[84] = ROL(key->rk[76] + delta[6][14], 1);
        key->rk[92] = ROL(key->rk[84] + delta[7][17], 6);
        key->rk[100] = ROL(key->rk[92] + delta[0][20], 13);
        key->rk[108] = ROL(key->rk[100] + delta[2][18], 1);
        key->rk[116] = ROL(key->rk[108] + delta[3][21], 6);
        key->rk[124] = ROL(key->rk[116] + delta[4][24], 13);
        key->rk[132] = ROL(key->rk[124] + delta[6][22], 1);
        key->rk[140] = ROL(key->rk[132] + delta[7][25], 6);
        key->rk[148] = ROL(key->rk[140] + delta[0][28], 13);
        key->rk[156] = ROL(key->rk[148] + delta[2][26], 1);
        key->rk[164] = ROL(key->rk[156] + delta[3][29], 6);
        key->rk[172] = ROL(key->rk[164] + delta[4][0], 13);
        key->rk[180] = ROL(key->rk[172] + delta[6][30], 1);
        key->rk[188] = ROL(key->rk[180] + delta[7][1], 6);

        key->rk[5] = ROL(*((unsigned int*)mk + 5) + delta[0][5], 17);
        key->rk[13] = ROL(key->rk[5] + delta[2][3], 3);
        key->rk[21] = ROL(key->rk[13] + delta[3][6], 11);
        key->rk[29] = ROL(key->rk[21] + delta[4][9], 17);
        key->rk[37] = ROL(key->rk[29] + delta[6][7], 3);
        key->rk[45] = ROL(key->rk[37] + delta[7][10], 11);
        key->rk[53] = ROL(key->rk[45] + delta[0][13], 17);
        key->rk[61] = ROL(key->rk[53] + delta[2][11], 3);
        key->rk[69] = ROL(key->rk[61] + delta[3][14], 11);
        key->rk[77] = ROL(key->rk[69] + delta[4][17], 17);
        key->rk[85] = ROL(key->rk[77] + delta[6][15], 3);
        key->rk[93] = ROL(key->rk[85] + delta[7][18], 11);
        key->rk[101] = ROL(key->rk[93] + delta[0][21], 17);
        key->rk[109] = ROL(key->rk[101] + delta[2][19], 3);
        key->rk[117] = ROL(key->rk[109] + delta[3][22], 11);
        key->rk[125] = ROL(key->rk[117] + delta[4][25], 17);
        key->rk[133] = ROL(key->rk[125] + delta[6][23], 3);
        key->rk[141] = ROL(key->rk[133] + delta[7][26], 11);
        key->rk[149] = ROL(key->rk[141] + delta[0][29], 17);
        key->rk[157] = ROL(key->rk[149] + delta[2][27], 3);
        key->rk[165] = ROL(key->rk[157] + delta[3][30], 11);
        key->rk[173] = ROL(key->rk[165] + delta[4][1], 17);
        key->rk[181] = ROL(key->rk[173] + delta[6][31], 3);
        key->rk[189] = ROL(key->rk[181] + delta[7][2], 11);

        key->rk[6] = ROL(*((unsigned int*)mk + 6) + delta[1][1], 1);
        key->rk[14] = ROL(key->rk[6] + delta[2][4], 6);
        key->rk[22] = ROL(key->rk[14] + delta[3][7], 13);
        key->rk[30] = ROL(key->rk[22] + delta[5][5], 1);
        key->rk[38] = ROL(key->rk[30] + delta[6][8], 6);
        key->rk[46] = ROL(key->rk[38] + delta[7][11], 13);
        key->rk[54] = ROL(key->rk[46] + delta[1][9], 1);
        key->rk[62] = ROL(key->rk[54] + delta[2][12], 6);
        key->rk[70] = ROL(key->rk[62] + delta[3][15], 13);
        key->rk[78] = ROL(key->rk[70] + delta[5][13], 1);
        key->rk[86] = ROL(key->rk[78] + delta[6][16], 6);
        key->rk[94] = ROL(key->rk[86] + delta[7][19], 13);
        key->rk[102] = ROL(key->rk[94] + delta[1][17], 1);
        key->rk[110] = ROL(key->rk[102] + delta[2][20], 6);
        key->rk[118] = ROL(key->rk[110] + delta[3][23], 13);
        key->rk[126] = ROL(key->rk[118] + delta[5][21], 1);
        key->rk[134] = ROL(key->rk[126] + delta[6][24], 6);
        key->rk[142] = ROL(key->rk[134] + delta[7][27], 13);
        key->rk[150] = ROL(key->rk[142] + delta[1][25], 1);
        key->rk[158] = ROL(key->rk[150] + delta[2][28], 6);
        key->rk[166] = ROL(key->rk[158] + delta[3][31], 13);
        key->rk[174] = ROL(key->rk[166] + delta[5][29], 1);
        key->rk[182] = ROL(key->rk[174] + delta[6][0], 6);
        key->rk[190] = ROL(key->rk[182] + delta[7][3], 13);

        key->rk[7] = ROL(*((unsigned int*)mk + 7) + delta[1][2], 3);
        key->rk[15] = ROL(key->rk[7] + delta[2][5], 11);
        key->rk[23] = ROL(key->rk[15] + delta[3][8], 17);
        key->rk[31] = ROL(key->rk[23] + delta[5][6], 3);
        key->rk[39] = ROL(key->rk[31] + delta[6][9], 11);
        key->rk[47] = ROL(key->rk[39] + delta[7][12], 17);
        key->rk[55] = ROL(key->rk[47] + delta[1][10], 3);
        key->rk[63] = ROL(key->rk[55] + delta[2][13], 11);
        key->rk[71] = ROL(key->rk[63] + delta[3][16], 17);
        key->rk[79] = ROL(key->rk[71] + delta[5][14], 3);
        key->rk[87] = ROL(key->rk[79] + delta[6][17], 11);
        key->rk[95] = ROL(key->rk[87] + delta[7][20], 17);
        key->rk[103] = ROL(key->rk[95] + delta[1][18], 3);
        key->rk[111] = ROL(key->rk[103] + delta[2][21], 11);
        key->rk[119] = ROL(key->rk[111] + delta[3][24], 17);
        key->rk[127] = ROL(key->rk[119] + delta[5][22], 3);
        key->rk[135] = ROL(key->rk[127] + delta[6][25], 11);
        key->rk[143] = ROL(key->rk[135] + delta[7][28], 17);
        key->rk[151] = ROL(key->rk[143] + delta[1][26], 3);
        key->rk[159] = ROL(key->rk[151] + delta[2][29], 11);
        key->rk[167] = ROL(key->rk[159] + delta[3][0], 17);
        key->rk[175] = ROL(key->rk[167] + delta[5][30], 3);
        key->rk[183] = ROL(key->rk[175] + delta[6][1], 11);
        key->rk[191] = ROL(key->rk[183] + delta[7][4], 17);

        break;

    default:
        return;
    }

    key->round = (mk_len >> 1) + 16;
}

#define XOR8x16(r, a, b)            \
   *((r)      ) = *((a)      ) ^ *((b)      ),   \
   *((r) + 0x1) = *((a) + 0x1) ^ *((b) + 0x1),   \
   *((r) + 0x2) = *((a) + 0x2) ^ *((b) + 0x2),   \
   *((r) + 0x3) = *((a) + 0x3) ^ *((b) + 0x3),   \
   *((r) + 0x4) = *((a) + 0x4) ^ *((b) + 0x4),   \
   *((r) + 0x5) = *((a) + 0x5) ^ *((b) + 0x5),   \
   *((r) + 0x6) = *((a) + 0x6) ^ *((b) + 0x6),   \
   *((r) + 0x7) = *((a) + 0x7) ^ *((b) + 0x7),   \
   *((r) + 0x8) = *((a) + 0x8) ^ *((b) + 0x8),   \
   *((r) + 0x9) = *((a) + 0x9) ^ *((b) + 0x9),   \
   *((r) + 0xa) = *((a) + 0xa) ^ *((b) + 0xa),   \
   *((r) + 0xb) = *((a) + 0xb) ^ *((b) + 0xb),   \
   *((r) + 0xc) = *((a) + 0xc) ^ *((b) + 0xc),   \
   *((r) + 0xd) = *((a) + 0xd) ^ *((b) + 0xd),   \
   *((r) + 0xe) = *((a) + 0xe) ^ *((b) + 0xe),   \
   *((r) + 0xf) = *((a) + 0xf) ^ *((b) + 0xf)

#define RSHIFT8x16_1(v)                        \
   (v)[15] = ((v)[15] >> 1) | ((v)[14] << 7),      \
   (v)[14] = ((v)[14] >> 1) | ((v)[13] << 7),      \
   (v)[13] = ((v)[13] >> 1) | ((v)[12] << 7),      \
   (v)[12] = ((v)[12] >> 1) | ((v)[11] << 7),      \
   (v)[11] = ((v)[11] >> 1) | ((v)[10] << 7),      \
   (v)[10] = ((v)[10] >> 1) | ((v)[ 9] << 7),      \
   (v)[ 9] = ((v)[ 9] >> 1) | ((v)[ 8] << 7),      \
   (v)[ 8] = ((v)[ 8] >> 1) | ((v)[ 7] << 7),      \
   (v)[ 7] = ((v)[ 7] >> 1) | ((v)[ 6] << 7),      \
   (v)[ 6] = ((v)[ 6] >> 1) | ((v)[ 5] << 7),      \
   (v)[ 5] = ((v)[ 5] >> 1) | ((v)[ 4] << 7),      \
   (v)[ 4] = ((v)[ 4] >> 1) | ((v)[ 3] << 7),      \
   (v)[ 3] = ((v)[ 3] >> 1) | ((v)[ 2] << 7),      \
   (v)[ 2] = ((v)[ 2] >> 1) | ((v)[ 1] << 7),      \
   (v)[ 1] = ((v)[ 1] >> 1) | ((v)[ 0] << 7),      \
   (v)[ 0] = ((v)[ 0] >> 1)

#define RSHIFT8x16_4(v)                        \
   (v)[15] = ((v)[15] >> 4) | ((v)[14] << 4),      \
   (v)[14] = ((v)[14] >> 4) | ((v)[13] << 4),      \
   (v)[13] = ((v)[13] >> 4) | ((v)[12] << 4),      \
   (v)[12] = ((v)[12] >> 4) | ((v)[11] << 4),      \
   (v)[11] = ((v)[11] >> 4) | ((v)[10] << 4),      \
   (v)[10] = ((v)[10] >> 4) | ((v)[ 9] << 4),      \
   (v)[ 9] = ((v)[ 9] >> 4) | ((v)[ 8] << 4),      \
   (v)[ 8] = ((v)[ 8] >> 4) | ((v)[ 7] << 4),      \
   (v)[ 7] = ((v)[ 7] >> 4) | ((v)[ 6] << 4),      \
   (v)[ 6] = ((v)[ 6] >> 4) | ((v)[ 5] << 4),      \
   (v)[ 5] = ((v)[ 5] >> 4) | ((v)[ 4] << 4),      \
   (v)[ 4] = ((v)[ 4] >> 4) | ((v)[ 3] << 4),      \
   (v)[ 3] = ((v)[ 3] >> 4) | ((v)[ 2] << 4),      \
   (v)[ 2] = ((v)[ 2] >> 4) | ((v)[ 1] << 4),      \
   (v)[ 1] = ((v)[ 1] >> 4) | ((v)[ 0] << 4),      \
   (v)[ 0] = ((v)[ 0] >> 4)

#define RSHIFT8x16_8(v)                     \
   (v)[15] = (v)[14],      \
   (v)[14] = (v)[13],      \
   (v)[13] = (v)[12],      \
   (v)[12] = (v)[11],      \
   (v)[11] = (v)[10],      \
   (v)[10] = (v)[ 9],      \
   (v)[ 9] = (v)[ 8],      \
   (v)[ 8] = (v)[ 7],      \
   (v)[ 7] = (v)[ 6],      \
   (v)[ 6] = (v)[ 5],      \
   (v)[ 5] = (v)[ 4],      \
   (v)[ 4] = (v)[ 3],      \
   (v)[ 3] = (v)[ 2],      \
   (v)[ 2] = (v)[ 1],      \
   (v)[ 1] = (v)[ 0],      \
   (v)[ 0] = 0

#define CPY8x16(d, s)                                 \
   *((unsigned int *)(d)) = *((unsigned int *)(s)),         \
   *((unsigned int *)(d) + 1) = *((unsigned int *)(s) + 1),   \
   *((unsigned int *)(d) + 2) = *((unsigned int *)(s) + 2),   \
   *((unsigned int *)(d) + 3) = *((unsigned int *)(s) + 3)

__host__ __device__  void lea_encrypt(uint8_t* ct, const uint8_t* pt, const LEA_KEY* key)
{
    unsigned int X0, X1, X2, X3;

    const unsigned int* _pt = (const unsigned int*)pt;
    unsigned int* _ct = (unsigned int*)ct;


    X0 = *(unsigned int*)(_pt + 0);
    X1 = *(unsigned int*)(_pt + 1);
    X2 = *(unsigned int*)(_pt + 2);
    X3 = *(unsigned int*)(_pt + 3);



    X3 = ROR((X2 ^ key->rk[4]) + (X3 ^ key->rk[5]), 3);
    X2 = ROR((X1 ^ key->rk[2]) + (X2 ^ key->rk[3]), 5);
    X1 = ROL((X0 ^ key->rk[0]) + (X1 ^ key->rk[1]), 9);
    X0 = ROR((X3 ^ key->rk[10]) + (X0 ^ key->rk[11]), 3);
    X3 = ROR((X2 ^ key->rk[8]) + (X3 ^ key->rk[9]), 5);
    X2 = ROL((X1 ^ key->rk[6]) + (X2 ^ key->rk[7]), 9);
    X1 = ROR((X0 ^ key->rk[16]) + (X1 ^ key->rk[17]), 3);
    X0 = ROR((X3 ^ key->rk[14]) + (X0 ^ key->rk[15]), 5);
    X3 = ROL((X2 ^ key->rk[12]) + (X3 ^ key->rk[13]), 9);
    X2 = ROR((X1 ^ key->rk[22]) + (X2 ^ key->rk[23]), 3);
    X1 = ROR((X0 ^ key->rk[20]) + (X1 ^ key->rk[21]), 5);
    X0 = ROL((X3 ^ key->rk[18]) + (X0 ^ key->rk[19]), 9);

    X3 = ROR((X2 ^ key->rk[28]) + (X3 ^ key->rk[29]), 3);
    X2 = ROR((X1 ^ key->rk[26]) + (X2 ^ key->rk[27]), 5);
    X1 = ROL((X0 ^ key->rk[24]) + (X1 ^ key->rk[25]), 9);
    X0 = ROR((X3 ^ key->rk[34]) + (X0 ^ key->rk[35]), 3);
    X3 = ROR((X2 ^ key->rk[32]) + (X3 ^ key->rk[33]), 5);
    X2 = ROL((X1 ^ key->rk[30]) + (X2 ^ key->rk[31]), 9);
    X1 = ROR((X0 ^ key->rk[40]) + (X1 ^ key->rk[41]), 3);
    X0 = ROR((X3 ^ key->rk[38]) + (X0 ^ key->rk[39]), 5);
    X3 = ROL((X2 ^ key->rk[36]) + (X3 ^ key->rk[37]), 9);
    X2 = ROR((X1 ^ key->rk[46]) + (X2 ^ key->rk[47]), 3);
    X1 = ROR((X0 ^ key->rk[44]) + (X1 ^ key->rk[45]), 5);
    X0 = ROL((X3 ^ key->rk[42]) + (X0 ^ key->rk[43]), 9);

    X3 = ROR((X2 ^ key->rk[52]) + (X3 ^ key->rk[53]), 3);
    X2 = ROR((X1 ^ key->rk[50]) + (X2 ^ key->rk[51]), 5);
    X1 = ROL((X0 ^ key->rk[48]) + (X1 ^ key->rk[49]), 9);
    X0 = ROR((X3 ^ key->rk[58]) + (X0 ^ key->rk[59]), 3);
    X3 = ROR((X2 ^ key->rk[56]) + (X3 ^ key->rk[57]), 5);
    X2 = ROL((X1 ^ key->rk[54]) + (X2 ^ key->rk[55]), 9);
    X1 = ROR((X0 ^ key->rk[64]) + (X1 ^ key->rk[65]), 3);
    X0 = ROR((X3 ^ key->rk[62]) + (X0 ^ key->rk[63]), 5);
    X3 = ROL((X2 ^ key->rk[60]) + (X3 ^ key->rk[61]), 9);
    X2 = ROR((X1 ^ key->rk[70]) + (X2 ^ key->rk[71]), 3);
    X1 = ROR((X0 ^ key->rk[68]) + (X1 ^ key->rk[69]), 5);
    X0 = ROL((X3 ^ key->rk[66]) + (X0 ^ key->rk[67]), 9);

    X3 = ROR((X2 ^ key->rk[76]) + (X3 ^ key->rk[77]), 3);
    X2 = ROR((X1 ^ key->rk[74]) + (X2 ^ key->rk[75]), 5);
    X1 = ROL((X0 ^ key->rk[72]) + (X1 ^ key->rk[73]), 9);
    X0 = ROR((X3 ^ key->rk[82]) + (X0 ^ key->rk[83]), 3);
    X3 = ROR((X2 ^ key->rk[80]) + (X3 ^ key->rk[81]), 5);
    X2 = ROL((X1 ^ key->rk[78]) + (X2 ^ key->rk[79]), 9);
    X1 = ROR((X0 ^ key->rk[88]) + (X1 ^ key->rk[89]), 3);
    X0 = ROR((X3 ^ key->rk[86]) + (X0 ^ key->rk[87]), 5);
    X3 = ROL((X2 ^ key->rk[84]) + (X3 ^ key->rk[85]), 9);
    X2 = ROR((X1 ^ key->rk[94]) + (X2 ^ key->rk[95]), 3);
    X1 = ROR((X0 ^ key->rk[92]) + (X1 ^ key->rk[93]), 5);
    X0 = ROL((X3 ^ key->rk[90]) + (X0 ^ key->rk[91]), 9);

    X3 = ROR((X2 ^ key->rk[100]) + (X3 ^ key->rk[101]), 3);
    X2 = ROR((X1 ^ key->rk[98]) + (X2 ^ key->rk[99]), 5);
    X1 = ROL((X0 ^ key->rk[96]) + (X1 ^ key->rk[97]), 9);
    X0 = ROR((X3 ^ key->rk[106]) + (X0 ^ key->rk[107]), 3);
    X3 = ROR((X2 ^ key->rk[104]) + (X3 ^ key->rk[105]), 5);
    X2 = ROL((X1 ^ key->rk[102]) + (X2 ^ key->rk[103]), 9);
    X1 = ROR((X0 ^ key->rk[112]) + (X1 ^ key->rk[113]), 3);
    X0 = ROR((X3 ^ key->rk[110]) + (X0 ^ key->rk[111]), 5);
    X3 = ROL((X2 ^ key->rk[108]) + (X3 ^ key->rk[109]), 9);
    X2 = ROR((X1 ^ key->rk[118]) + (X2 ^ key->rk[119]), 3);
    X1 = ROR((X0 ^ key->rk[116]) + (X1 ^ key->rk[117]), 5);
    X0 = ROL((X3 ^ key->rk[114]) + (X0 ^ key->rk[115]), 9);

    X3 = ROR((X2 ^ key->rk[124]) + (X3 ^ key->rk[125]), 3);
    X2 = ROR((X1 ^ key->rk[122]) + (X2 ^ key->rk[123]), 5);
    X1 = ROL((X0 ^ key->rk[120]) + (X1 ^ key->rk[121]), 9);
    X0 = ROR((X3 ^ key->rk[130]) + (X0 ^ key->rk[131]), 3);
    X3 = ROR((X2 ^ key->rk[128]) + (X3 ^ key->rk[129]), 5);
    X2 = ROL((X1 ^ key->rk[126]) + (X2 ^ key->rk[127]), 9);
    X1 = ROR((X0 ^ key->rk[136]) + (X1 ^ key->rk[137]), 3);
    X0 = ROR((X3 ^ key->rk[134]) + (X0 ^ key->rk[135]), 5);
    X3 = ROL((X2 ^ key->rk[132]) + (X3 ^ key->rk[133]), 9);
    X2 = ROR((X1 ^ key->rk[142]) + (X2 ^ key->rk[143]), 3);
    X1 = ROR((X0 ^ key->rk[140]) + (X1 ^ key->rk[141]), 5);
    X0 = ROL((X3 ^ key->rk[138]) + (X0 ^ key->rk[139]), 9);

    if (key->round > 24)
    {
        X3 = ROR((X2 ^ key->rk[148]) + (X3 ^ key->rk[149]), 3);
        X2 = ROR((X1 ^ key->rk[146]) + (X2 ^ key->rk[147]), 5);
        X1 = ROL((X0 ^ key->rk[144]) + (X1 ^ key->rk[145]), 9);
        X0 = ROR((X3 ^ key->rk[154]) + (X0 ^ key->rk[155]), 3);
        X3 = ROR((X2 ^ key->rk[152]) + (X3 ^ key->rk[153]), 5);
        X2 = ROL((X1 ^ key->rk[150]) + (X2 ^ key->rk[151]), 9);
        X1 = ROR((X0 ^ key->rk[160]) + (X1 ^ key->rk[161]), 3);
        X0 = ROR((X3 ^ key->rk[158]) + (X0 ^ key->rk[159]), 5);
        X3 = ROL((X2 ^ key->rk[156]) + (X3 ^ key->rk[157]), 9);
        X2 = ROR((X1 ^ key->rk[166]) + (X2 ^ key->rk[167]), 3);
        X1 = ROR((X0 ^ key->rk[164]) + (X1 ^ key->rk[165]), 5);
        X0 = ROL((X3 ^ key->rk[162]) + (X0 ^ key->rk[163]), 9);
    }
    if (key->round > 28)
    {
        X3 = ROR((X2 ^ key->rk[172]) + (X3 ^ key->rk[173]), 3);
        X2 = ROR((X1 ^ key->rk[170]) + (X2 ^ key->rk[171]), 5);
        X1 = ROL((X0 ^ key->rk[168]) + (X1 ^ key->rk[169]), 9);
        X0 = ROR((X3 ^ key->rk[178]) + (X0 ^ key->rk[179]), 3);
        X3 = ROR((X2 ^ key->rk[176]) + (X3 ^ key->rk[177]), 5);
        X2 = ROL((X1 ^ key->rk[174]) + (X2 ^ key->rk[175]), 9);
        X1 = ROR((X0 ^ key->rk[184]) + (X1 ^ key->rk[185]), 3);
        X0 = ROR((X3 ^ key->rk[182]) + (X0 ^ key->rk[183]), 5);
        X3 = ROL((X2 ^ key->rk[180]) + (X3 ^ key->rk[181]), 9);
        X2 = ROR((X1 ^ key->rk[190]) + (X2 ^ key->rk[191]), 3);
        X1 = ROR((X0 ^ key->rk[188]) + (X1 ^ key->rk[189]), 5);
        X0 = ROL((X3 ^ key->rk[186]) + (X0 ^ key->rk[187]), 9);
    }
    *(unsigned int*)(_ct + 0) = X0;
    *(unsigned int*)(_ct + 1) = X1;
    *(unsigned int*)(_ct + 2) = X2;
    *(unsigned int*)(_ct + 3) = X3;
}
//진짜 곱셈
__host__ __device__  static void gcm_gfmul_m(uint8_t* r, const uint8_t* x, const uint8_t* y)
{
    uint8_t z[16] = { 0x00, };
    uint8_t v[16] = { 0x00, };
    int i = 0;
    memcpy(v, y, 16);

    for (i = 0; i < 128; i++)
    {
        if ((x[i >> 3] >> (7 - (i & 0x7))) & 1) {
            //printf("%d , x[i >> 3] >> (7 - (i & 0x7))) & 1 : %d\n", i, ((x[i >> 3] >> (7 - (i & 0x7))) & 1));
            XOR8x16(z, z, v);
        }
        if (v[15] & 1)
        {
            RSHIFT8x16_1(v);

            v[0] ^= 0xe1;
        }
        else
            RSHIFT8x16_1(v);

    }

    memcpy(r, z, 16);
}
__host__ __device__  static void _gcm_ghash_m(uint8_t* r, const uint8_t* x, int x_len, const uint8_t h[][16])
{
    int i;
    uint8_t y[16] = { 0, };

    memcpy(y, r, 16);

    for (; x_len >= 16; x_len -= 16, x += 16)
    {
        XOR8x16(y, y, x);

        gcm_gfmul_m(y, y, h[0]);
    }

    if (x_len)
    {
        for (i = 0; i < x_len; i++)
            y[i] = y[i] ^ x[i];

        gcm_gfmul_m(y, y, h[0]);
    }

    memcpy(r, y, 16);
}
__host__ __device__  static void _lea_gcm_init_m(LEA_GCM_CTX* ctx, const uint8_t* mk, int mk_len)//ctx->h[0] = H (GHASH에서 쓰는 값)
{
    uint8_t zero[16] = { 0, };

    memset(ctx, 0, sizeof(LEA_GCM_CTX));

    lea_set_key(&ctx->key, mk, mk_len);
    lea_encrypt((uint8_t*)ctx->h, zero, &ctx->key);
}
//4bit table
__constant__ const unsigned char reduction_4bit[16][2] = {
    { 0x00, 0x00 }, { 0x1c, 0x20 }, { 0x38, 0x40 }, { 0x24, 0x60 }, { 0x70, 0x80 }, { 0x6c, 0xa0 }, { 0x48, 0xc0 }, { 0x54, 0xe0 },
    { 0xe1, 0x00 }, { 0xfd, 0x20 }, { 0xd9, 0x40 }, { 0xc5, 0x60 }, { 0x91, 0x80 }, { 0x8d, 0xa0 }, { 0xa9, 0xc0 }, { 0xb5, 0xe0 }
};
__host__ __device__  static void gcm_init_4bit_table(unsigned char hTable[][16], const unsigned char* h)
{
    unsigned char tmp[16];

    memcpy(tmp, h, 16);
    ;
    memcpy(hTable[8], tmp, 16);

    RSHIFT8x16_1(tmp);
    if (hTable[8][15] & 1)
        tmp[0] ^= 0xe1;
    memcpy(hTable[4], tmp, 16);

    RSHIFT8x16_1(tmp);
    if (hTable[4][15] & 1)
        tmp[0] ^= 0xe1;
    memcpy(hTable[2], tmp, 16);


    RSHIFT8x16_1(tmp);
    if (hTable[2][15] & 1)
        tmp[0] ^= 0xe1;
    memcpy(hTable[1], tmp, 16);


    XOR8x16(hTable[3], hTable[2], hTable[1]);
    XOR8x16(hTable[5], hTable[4], hTable[1]);
    XOR8x16(hTable[6], hTable[4], hTable[2]);
    XOR8x16(hTable[7], hTable[4], hTable[3]);
    XOR8x16(hTable[9], hTable[8], hTable[1]);
    XOR8x16(hTable[10], hTable[8], hTable[2]);
    XOR8x16(hTable[11], hTable[8], hTable[3]);
    XOR8x16(hTable[12], hTable[8], hTable[4]);
    XOR8x16(hTable[13], hTable[8], hTable[5]);
    XOR8x16(hTable[14], hTable[8], hTable[6]);
    XOR8x16(hTable[15], hTable[8], hTable[7]);
}
__host__ __device__  static void gcm_gfmul_4(unsigned char* r, const unsigned char* x, const unsigned char hTable[][16])
{
    unsigned char z[16], mask;
    int i;

    memset(z, 0, 16);

    for (i = 15; i > 0; i--)
    {
        mask = x[i] & 0xf;
        XOR8x16(z, z, hTable[mask]);

        mask = z[15] & 0xf;
        RSHIFT8x16_4(z);
        z[0] ^= reduction_4bit[mask][0];
        z[1] ^= reduction_4bit[mask][1];
        ////======
        mask = x[i] >> 4;
        XOR8x16(z, z, hTable[mask]);

        mask = z[15] & 0xf;
        RSHIFT8x16_4(z);
        z[0] ^= reduction_4bit[mask][0];
        z[1] ^= reduction_4bit[mask][1];

    }

    mask = x[i] & 0xf;
    XOR8x16(z, z, hTable[mask]);

    mask = z[15] & 0xf;
    RSHIFT8x16_4(z);
    z[0] ^= reduction_4bit[mask][0];
    z[1] ^= reduction_4bit[mask][1];

    mask = x[i] >> 4;
    XOR8x16(z, z, hTable[mask]);

    memcpy(r, z, 16);
}
__host__ __device__  static void _gcm_ghash_4(unsigned char* r, const unsigned char* x, unsigned int x_len, const unsigned char hTable[][16])
{
    unsigned int i;
    unsigned char y[16] = { 0, };

    memcpy(y, r, 16);

    for (; x_len >= 16; x_len -= 16, x += 16)
    {
        XOR8x16(y, y, x);

        gcm_gfmul_4(y, y, hTable);
    }

    if (x_len)
    {
        for (i = 0; i < x_len; i++)
            y[i] = y[i] ^ x[i];

        gcm_gfmul_4(y, y, hTable);
    }

    memcpy(r, y, 16);
}
__host__ __device__  static void _lea_gcm_init_4(LEA_GCM_CTX* ctx, const unsigned char* mk, int mk_len)
{
    unsigned char zero[16] = { 0, }, h[16] = { 0x00, };

    memset(ctx, 0, sizeof(LEA_GCM_CTX));

    lea_set_key(&ctx->key, mk, mk_len);
    lea_encrypt(zero, zero, &ctx->key);

    gcm_init_4bit_table(ctx->h, zero);
}
// 8bit 테이블 버전
__constant__ const unsigned char reduction_8bit[256][2] = {
    { 0x00, 0x00 }, { 0x01, 0xc2 }, { 0x03, 0x84 }, { 0x02, 0x46 }, { 0x07, 0x08 }, { 0x06, 0xca }, { 0x04, 0x8c }, { 0x05, 0x4e },
    { 0x0e, 0x10 }, { 0x0f, 0xd2 }, { 0x0d, 0x94 }, { 0x0c, 0x56 }, { 0x09, 0x18 }, { 0x08, 0xda }, { 0x0a, 0x9c }, { 0x0b, 0x5e },
    { 0x1c, 0x20 }, { 0x1d, 0xe2 }, { 0x1f, 0xa4 }, { 0x1e, 0x66 }, { 0x1b, 0x28 }, { 0x1a, 0xea }, { 0x18, 0xac }, { 0x19, 0x6e },
    { 0x12, 0x30 }, { 0x13, 0xf2 }, { 0x11, 0xb4 }, { 0x10, 0x76 }, { 0x15, 0x38 }, { 0x14, 0xfa }, { 0x16, 0xbc }, { 0x17, 0x7e },
    { 0x38, 0x40 }, { 0x39, 0x82 }, { 0x3b, 0xc4 }, { 0x3a, 0x06 }, { 0x3f, 0x48 }, { 0x3e, 0x8a }, { 0x3c, 0xcc }, { 0x3d, 0x0e },
    { 0x36, 0x50 }, { 0x37, 0x92 }, { 0x35, 0xd4 }, { 0x34, 0x16 }, { 0x31, 0x58 }, { 0x30, 0x9a }, { 0x32, 0xdc }, { 0x33, 0x1e },
    { 0x24, 0x60 }, { 0x25, 0xa2 }, { 0x27, 0xe4 }, { 0x26, 0x26 }, { 0x23, 0x68 }, { 0x22, 0xaa }, { 0x20, 0xec }, { 0x21, 0x2e },
    { 0x2a, 0x70 }, { 0x2b, 0xb2 }, { 0x29, 0xf4 }, { 0x28, 0x36 }, { 0x2d, 0x78 }, { 0x2c, 0xba }, { 0x2e, 0xfc }, { 0x2f, 0x3e },
    { 0x70, 0x80 }, { 0x71, 0x42 }, { 0x73, 0x04 }, { 0x72, 0xc6 }, { 0x77, 0x88 }, { 0x76, 0x4a }, { 0x74, 0x0c }, { 0x75, 0xce },
    { 0x7e, 0x90 }, { 0x7f, 0x52 }, { 0x7d, 0x14 }, { 0x7c, 0xd6 }, { 0x79, 0x98 }, { 0x78, 0x5a }, { 0x7a, 0x1c }, { 0x7b, 0xde },
    { 0x6c, 0xa0 }, { 0x6d, 0x62 }, { 0x6f, 0x24 }, { 0x6e, 0xe6 }, { 0x6b, 0xa8 }, { 0x6a, 0x6a }, { 0x68, 0x2c }, { 0x69, 0xee },
    { 0x62, 0xb0 }, { 0x63, 0x72 }, { 0x61, 0x34 }, { 0x60, 0xf6 }, { 0x65, 0xb8 }, { 0x64, 0x7a }, { 0x66, 0x3c }, { 0x67, 0xfe },
    { 0x48, 0xc0 }, { 0x49, 0x02 }, { 0x4b, 0x44 }, { 0x4a, 0x86 }, { 0x4f, 0xc8 }, { 0x4e, 0x0a }, { 0x4c, 0x4c }, { 0x4d, 0x8e },
    { 0x46, 0xd0 }, { 0x47, 0x12 }, { 0x45, 0x54 }, { 0x44, 0x96 }, { 0x41, 0xd8 }, { 0x40, 0x1a }, { 0x42, 0x5c }, { 0x43, 0x9e },
    { 0x54, 0xe0 }, { 0x55, 0x22 }, { 0x57, 0x64 }, { 0x56, 0xa6 }, { 0x53, 0xe8 }, { 0x52, 0x2a }, { 0x50, 0x6c }, { 0x51, 0xae },
    { 0x5a, 0xf0 }, { 0x5b, 0x32 }, { 0x59, 0x74 }, { 0x58, 0xb6 }, { 0x5d, 0xf8 }, { 0x5c, 0x3a }, { 0x5e, 0x7c }, { 0x5f, 0xbe },
    { 0xe1, 0x00 }, { 0xe0, 0xc2 }, { 0xe2, 0x84 }, { 0xe3, 0x46 }, { 0xe6, 0x08 }, { 0xe7, 0xca }, { 0xe5, 0x8c }, { 0xe4, 0x4e },
    { 0xef, 0x10 }, { 0xee, 0xd2 }, { 0xec, 0x94 }, { 0xed, 0x56 }, { 0xe8, 0x18 }, { 0xe9, 0xda }, { 0xeb, 0x9c }, { 0xea, 0x5e },
    { 0xfd, 0x20 }, { 0xfc, 0xe2 }, { 0xfe, 0xa4 }, { 0xff, 0x66 }, { 0xfa, 0x28 }, { 0xfb, 0xea }, { 0xf9, 0xac }, { 0xf8, 0x6e },
    { 0xf3, 0x30 }, { 0xf2, 0xf2 }, { 0xf0, 0xb4 }, { 0xf1, 0x76 }, { 0xf4, 0x38 }, { 0xf5, 0xfa }, { 0xf7, 0xbc }, { 0xf6, 0x7e },
    { 0xd9, 0x40 }, { 0xd8, 0x82 }, { 0xda, 0xc4 }, { 0xdb, 0x06 }, { 0xde, 0x48 }, { 0xdf, 0x8a }, { 0xdd, 0xcc }, { 0xdc, 0x0e },
    { 0xd7, 0x50 }, { 0xd6, 0x92 }, { 0xd4, 0xd4 }, { 0xd5, 0x16 }, { 0xd0, 0x58 }, { 0xd1, 0x9a }, { 0xd3, 0xdc }, { 0xd2, 0x1e },
    { 0xc5, 0x60 }, { 0xc4, 0xa2 }, { 0xc6, 0xe4 }, { 0xc7, 0x26 }, { 0xc2, 0x68 }, { 0xc3, 0xaa }, { 0xc1, 0xec }, { 0xc0, 0x2e },
    { 0xcb, 0x70 }, { 0xca, 0xb2 }, { 0xc8, 0xf4 }, { 0xc9, 0x36 }, { 0xcc, 0x78 }, { 0xcd, 0xba }, { 0xcf, 0xfc }, { 0xce, 0x3e },
    { 0x91, 0x80 }, { 0x90, 0x42 }, { 0x92, 0x04 }, { 0x93, 0xc6 }, { 0x96, 0x88 }, { 0x97, 0x4a }, { 0x95, 0x0c }, { 0x94, 0xce },
    { 0x9f, 0x90 }, { 0x9e, 0x52 }, { 0x9c, 0x14 }, { 0x9d, 0xd6 }, { 0x98, 0x98 }, { 0x99, 0x5a }, { 0x9b, 0x1c }, { 0x9a, 0xde },
    { 0x8d, 0xa0 }, { 0x8c, 0x62 }, { 0x8e, 0x24 }, { 0x8f, 0xe6 }, { 0x8a, 0xa8 }, { 0x8b, 0x6a }, { 0x89, 0x2c }, { 0x88, 0xee },
    { 0x83, 0xb0 }, { 0x82, 0x72 }, { 0x80, 0x34 }, { 0x81, 0xf6 }, { 0x84, 0xb8 }, { 0x85, 0x7a }, { 0x87, 0x3c }, { 0x86, 0xfe },
    { 0xa9, 0xc0 }, { 0xa8, 0x02 }, { 0xaa, 0x44 }, { 0xab, 0x86 }, { 0xae, 0xc8 }, { 0xaf, 0x0a }, { 0xad, 0x4c }, { 0xac, 0x8e },
    { 0xa7, 0xd0 }, { 0xa6, 0x12 }, { 0xa4, 0x54 }, { 0xa5, 0x96 }, { 0xa0, 0xd8 }, { 0xa1, 0x1a }, { 0xa3, 0x5c }, { 0xa2, 0x9e },
    { 0xb5, 0xe0 }, { 0xb4, 0x22 }, { 0xb6, 0x64 }, { 0xb7, 0xa6 }, { 0xb2, 0xe8 }, { 0xb3, 0x2a }, { 0xb1, 0x6c }, { 0xb0, 0xae },
    { 0xbb, 0xf0 }, { 0xba, 0x32 }, { 0xb8, 0x74 }, { 0xb9, 0xb6 }, { 0xbc, 0xf8 }, { 0xbd, 0x3a }, { 0xbf, 0x7c }, { 0xbe, 0xbe },
};
__host__ __device__  static void gcm_init_8bit_table(unsigned char hTable[][16], const unsigned char* h)
{
    unsigned char tmp[16];
    unsigned int i, j;

    memcpy(tmp, h, 16);
    memcpy(hTable[0x80], tmp, 16);

    for (i = 0x40; i >= 1; i >>= 1)
    {
        RSHIFT8x16_1(tmp);
        if (hTable[i << 1][15] & 1)
            tmp[0] ^= 0xe1;
        memcpy(hTable[i], tmp, 16);
    }

    for (i = 2; i < 256; i <<= 1)
    {
        for (j = 1; j < i; j++)
            XOR8x16(hTable[i + j], hTable[i], hTable[j]);
    }
}
__host__ __device__  static void gcm_gfmul_8(unsigned char* r, const unsigned char* x, const unsigned char hTable[][16])
{
    unsigned char z[16], mask;
    int i;

    memset(z, 0, 16);

    for (i = 15; i > 0; i--)
    {
        XOR8x16(z, z, hTable[x[i]]);

        mask = z[15];
        RSHIFT8x16_8(z);
        z[0] ^= reduction_8bit[mask][0];
        z[1] ^= reduction_8bit[mask][1];
    }

    XOR8x16(z, z, hTable[x[i]]);

    memcpy(r, z, 16);
}
__host__ __device__  static void _gcm_ghash_8(unsigned char* r, const unsigned char* x, unsigned int x_len, const unsigned char hTable[][16])
{
    unsigned int i;
    unsigned char y[16] = { 0, };

    memcpy(y, r, 16);

    for (; x_len >= 16; x_len -= 16, x += 16)
    {
        XOR8x16(y, y, x);

        gcm_gfmul_8(y, y, hTable);
    }

    if (x_len)
    {
        for (i = 0; i < x_len; i++)
            y[i] = y[i] ^ x[i];

        gcm_gfmul_8(y, y, hTable);
    }

    memcpy(r, y, 16);
}
__host__ __device__  static void _lea_gcm_init_8(LEA_GCM_CTX* ctx, const unsigned char* mk, int mk_len)
{
    unsigned char zero[16] = { 0, };

    memset(ctx, 0, sizeof(LEA_GCM_CTX));

    lea_set_key(&ctx->key, mk, mk_len);
    lea_encrypt(zero, zero, &ctx->key);

    gcm_init_8bit_table(ctx->h, zero);
}

//내 코드
__device__ void parallel_ghash1(uint8_t* dest, uint8_t* src1, uint8_t* src2, uint8_t* H_8) {
    uint8_t temp[16] = { 0x00, };
    gcm_gfmul_m(temp, src1, H_8);//->X_i*H^8
    XOR8x16(dest, temp, src2);//X_i *H^8  + X_(i+8)
    // __syncthreads();
}
__device__ void parallel_ghash2(uint8_t* dest, uint8_t* src1, uint8_t* src2, uint8_t* H_4) {
    uint8_t temp[16] = { 0x00, };
    gcm_gfmul_m(temp, src1, H_4);//->X_i*H^4
    XOR8x16(dest, temp, src2);//X_i *H^8  + X_(i+4)
    // __syncthreads();
}
__device__ void parallel_ghash3(uint8_t* dest, uint8_t* src1, uint8_t* src2, uint8_t* H_2) {
    uint8_t temp[16] = { 0x00, };
    gcm_gfmul_m(temp, src1, H_2);//->X_i*H^2
    XOR8x16(dest, temp, src2);//X_i *H^8  + X_(i+4)
}
__device__ void parallel_ghash_last(uint8_t* dest, uint8_t* src1, uint8_t* src2, uint8_t* H_2, uint8_t* H_1) {
    uint8_t temp1[16] = { 0x00, };
    uint8_t temp2[16] = { 0x00, };
    gcm_gfmul_m(temp1, src1, H_2);//->X_i*H^2
    gcm_gfmul_m(temp2, src2, H_1);//->X_i*H
    XOR8x16(dest, temp1, temp2);//X_i *H^8  + X_(i+4)
}
//GCM_REF를 위한 함수들임
__host__ __device__ static void ctr128_inc(unsigned char* counter) {
    unsigned int n = 16;
    unsigned char c;

    do {
        --n;
        c = counter[n];
        ++c;
        counter[n] = c;
        if (c) return;
    } while (n);
}
__host__ __device__ static void ctr128_inc_aligned(unsigned char* counter) {
    unsigned int* data, c, n;
    const union { long one; char little; } is_endian = { 1 };

    if (is_endian.little) {
        ctr128_inc(counter);
        return;
    }

    data = (unsigned int*)counter;
    n = 16 / sizeof(unsigned int);
    do {
        --n;
        c = data[n];
        ++c;
        data[n] = c;
        if (c) return;
    } while (n);
}
__host__ __device__ void ctr_enc(unsigned char* ct, const unsigned char* pt, unsigned int pt_len, unsigned char* ctr, const LEA_KEY* key)
{
    unsigned char block[16] = { 0x00, };

    if (!ctr || !key || pt_len == 0) {
        return;
    }

    unsigned int numBlocks = pt_len >> 4;

    for (unsigned int i = 0; i < numBlocks; i++, pt += 16, ct += 16) {
        lea_encrypt(block, ctr, key);
        XOR8x16(ct, block, pt);
        ctr128_inc_aligned(ctr);
    }

    if (pt_len & 0xF) {
        lea_encrypt(block, ctr, key);
        for (unsigned int i = 0; i < (pt_len & 0xF); i++) {
            ct[i] = block[i] ^ pt[i];
        }
    }
}
__host__ __device__ void ctr_dec(unsigned char* pt, const unsigned char* ct, unsigned int ct_len, unsigned char* ctr, const LEA_KEY* key)
{
    ctr_enc(pt, ct, ct_len, ctr, key);
}
__host__ __device__ void gcm_set_ctr(LEA_GCM_CTX* ctx, const unsigned char* iv, int iv_len)
{
    int tmp_iv_len = iv_len;

    if (!ctx || !iv) {
        return;
    }
    if (iv_len < 0) {
        return;
    }

    ctx->ct_len = 0;

    if (iv_len == 12)
    {
        memcpy(ctx->ctr, iv, 12);
        ctx->ctr[15] = 1;
    }
    else
    {
        for (; iv_len >= 16; iv_len -= 16, iv += 16)
            _gcm_ghash_m(ctx->ctr, iv, 16, (const unsigned char(*)[16])ctx->h);

        if (iv_len)
            _gcm_ghash_m(ctx->ctr, iv, iv_len, (const unsigned char(*)[16])ctx->h);

        tmp_iv_len <<= 3;
        ctx->yn[12] = (tmp_iv_len >> 24) & 0xff;
        ctx->yn[13] = (tmp_iv_len >> 16) & 0xff;
        ctx->yn[14] = (tmp_iv_len >> 8) & 0xff;
        ctx->yn[15] = (tmp_iv_len) & 0xff;
        _gcm_ghash_m(ctx->ctr, ctx->yn, 16, (const unsigned char(*)[16])ctx->h);
        memset(ctx->yn, 0, 16);
    }

    lea_encrypt(ctx->ek0, ctx->ctr, &ctx->key);

    ctr128_inc_aligned(ctx->ctr);
}
__host__ __device__ void gcm_set_aad(LEA_GCM_CTX* ctx, const unsigned char* aad, int aad_len)
{
    if (!ctx) {
        return;
    }
    if (aad_len <= 0) {
        return;
    }
    if (!aad) {
        return;
    }
    ctx->aad_len = aad_len;

    _gcm_ghash_m(ctx->tbl, aad, aad_len, (const unsigned char(*)[16])ctx->h);
}
__host__ __device__ void gcm_enc(LEA_GCM_CTX* ctx, unsigned char* ct, const unsigned char* pt, int pt_len)
{
    int remain, i;

    if (!ctx || !ct || !pt) {
        return;
    }
    if (pt_len < 0) {
        return;
    }

    ctx->is_encrypt = 1;
    ctx->ct_len += pt_len;

    if (!pt_len)
        return;

    if (ctx->yn_used)
    {
        remain = 16 - ctx->yn_used;

        if (remain > pt_len)
            remain = pt_len;

        for (i = 0; i < remain; i++)
            ctx->yn[ctx->yn_used + i] ^= pt[i];

        memcpy(ct, ctx->yn + ctx->yn_used, remain);

        pt_len -= remain;
        pt += remain;
        ct += remain;
        ctx->yn_used = (ctx->yn_used + remain) & 0xf;

        if (!ctx->yn_used) {
            _gcm_ghash_m(ctx->tbl, ctx->yn, 16, (const unsigned char(*)[16])ctx->h);
        }


        if (!pt_len)
            return;
    }

    i = pt_len & 0xfffffff0;

    ctr_enc(ct, pt, i, ctx->ctr, &ctx->key);

    if (i) {
        _gcm_ghash_m(ctx->tbl, ct, i, (const unsigned char(*)[16])ctx->h);
    }


    pt_len &= 0xf;

    if (!pt_len)
        return;

    pt += i;
    ct += i;

    lea_encrypt(ctx->yn, ctx->ctr, &ctx->key);
    ctr128_inc_aligned(ctx->ctr);
    ctx->yn_used = pt_len;

    for (pt_len--; pt_len >= 0; pt_len--)
        ct[pt_len] = ctx->yn[pt_len] = ctx->yn[pt_len] ^ pt[pt_len];
}
__host__ __device__ int gcm_final(LEA_GCM_CTX* ctx, unsigned char* tag, int tag_len)
{
    unsigned char tmp[16];

    if (!ctx || !tag) {
        memset(ctx, 0, sizeof(LEA_GCM_CTX));
        return -1;
    }
    if (tag_len < 4) {
        memset(ctx, 0, sizeof(LEA_GCM_CTX));
        return -1;
    }

    if (ctx->yn_used)
        _gcm_ghash_m(ctx->tbl, ctx->yn, ctx->yn_used, (const unsigned char(*)[16])ctx->h);

    memset(tmp, 0, 16);

    ctx->aad_len <<= 3;
    ctx->ct_len <<= 3;

    tmp[4] = ctx->aad_len >> 24;
    tmp[5] = ctx->aad_len >> 16;
    tmp[6] = ctx->aad_len >> 8;
    tmp[7] = ctx->aad_len;

    tmp[12] = ctx->ct_len >> 24;
    tmp[13] = ctx->ct_len >> 16;
    tmp[14] = ctx->ct_len >> 8;
    tmp[15] = ctx->ct_len;
    _gcm_ghash_m(ctx->tbl, tmp, 16, (const unsigned char(*)[16])ctx->h);
    XOR8x16(ctx->tbl, ctx->tbl, ctx->ek0);
    if (ctx->is_encrypt)
        memcpy(tag, ctx->tbl, tag_len);
    else
    {
        for (tag_len--; tag_len >= 0; tag_len--)
        {
            if (ctx->tbl[tag_len] != tag[tag_len]) {
                memset(ctx->ctr, 0, 16);
                memset(ctx->ek0, 0, 16);
                memset(ctx->tbl, 0, 16);
                memset(ctx->yn, 0, 16);
                ctx->yn_used = 0;

                return -1;

            }

        }
    }
    ctx->ct_len = 0;

    memset(ctx->ctr, 0, 16);
    memset(ctx->ek0, 0, 16);
    memset(ctx->tbl, 0, 16);
    memset(ctx->yn, 0, 16);
    ctx->yn_used = 0;

    return 0;
}

//4bit table의 GCM
__host__ __device__ void gcm_set_ctr_4(LEA_GCM_CTX* ctx, const unsigned char* iv, int iv_len)
{

    int tmp_iv_len = iv_len;

    if (!ctx || !iv) {
        return;
    }
    if (iv_len < 0) {
        return;
    }

    ctx->ct_len = 0;

    if (iv_len == 12)
    {
        memcpy(ctx->ctr, iv, 12);
        ctx->ctr[15] = 1;
    }
    else
    {
        for (; iv_len >= 16; iv_len -= 16, iv += 16)
            _gcm_ghash_4(ctx->ctr, iv, 16, (const unsigned char(*)[16])ctx->h);

        if (iv_len)
            _gcm_ghash_4(ctx->ctr, iv, iv_len, (const unsigned char(*)[16])ctx->h);

        tmp_iv_len <<= 3;
        ctx->yn[12] = (tmp_iv_len >> 24) & 0xff;
        ctx->yn[13] = (tmp_iv_len >> 16) & 0xff;
        ctx->yn[14] = (tmp_iv_len >> 8) & 0xff;
        ctx->yn[15] = (tmp_iv_len) & 0xff;
        _gcm_ghash_4(ctx->ctr, ctx->yn, 16, (const unsigned char(*)[16])ctx->h);
        memset(ctx->yn, 0, 16);
    }

    lea_encrypt(ctx->ek0, ctx->ctr, &ctx->key);

    ctr128_inc_aligned(ctx->ctr);
}
__host__ __device__ void gcm_set_aad_4(LEA_GCM_CTX* ctx, const unsigned char* aad, int aad_len)
{
    if (!ctx) {
        return;
    }
    if (!aad) {
        return;
    }
    if (aad_len <= 0) {
        return;
    }

    ctx->aad_len = aad_len;


    _gcm_ghash_4(ctx->tbl, aad, aad_len, (const unsigned char(*)[16])ctx->h);
}
__host__ __device__ void gcm_enc_4(LEA_GCM_CTX* ctx, unsigned char* ct, const unsigned char* pt, int pt_len)
{
    int remain, i;

    if (!ctx || !ct || !pt) {
        return;
    }
    if (pt_len < 0) {
        return;
    }

    ctx->is_encrypt = 1;
    ctx->ct_len += pt_len;

    if (!pt_len)
        return;

    if (ctx->yn_used)
    {
        remain = 16 - ctx->yn_used;

        if (remain > pt_len)
            remain = pt_len;

        for (i = 0; i < remain; i++)
            ctx->yn[ctx->yn_used + i] ^= pt[i];

        memcpy(ct, ctx->yn + ctx->yn_used, remain);

        pt_len -= remain;
        pt += remain;
        ct += remain;
        ctx->yn_used = (ctx->yn_used + remain) & 0xf;

        if (!ctx->yn_used)
            _gcm_ghash_4(ctx->tbl, ctx->yn, 16, (const unsigned char(*)[16])ctx->h);

        if (!pt_len)
            return;
    }

    i = pt_len & 0xfffffff0;

    ctr_enc(ct, pt, i, ctx->ctr, &ctx->key);

    if (i)
        _gcm_ghash_4(ctx->tbl, ct, i, (const unsigned char(*)[16])ctx->h);

    pt_len &= 0xf;

    if (!pt_len)
        return;

    pt += i;
    ct += i;

    lea_encrypt(ctx->yn, ctx->ctr, &ctx->key);
    ctr128_inc_aligned(ctx->ctr);
    ctx->yn_used = pt_len;

    for (pt_len--; pt_len >= 0; pt_len--)
        ct[pt_len] = ctx->yn[pt_len] = ctx->yn[pt_len] ^ pt[pt_len];
}
__host__ __device__ void gcm_dec_4(LEA_GCM_CTX* ctx, unsigned char* pt, const unsigned char* ct, int ct_len)
{
    int remain, i;

    if (!ctx || !pt || !ct) {
        return;
    }
    if (ct_len < 0) {
        return;
    }

    ctx->is_encrypt = 0;
    ctx->ct_len += ct_len;

    if (!ct_len)
        return;

    if (ctx->yn_used)
    {
        remain = 16 - ctx->yn_used;

        if (remain > ct_len)
            remain = ct_len;

        for (i = 0; i < remain; i++)
            pt[i] = ctx->yn[ctx->yn_used + i] ^ ct[i];

        memcpy(ctx->yn + ctx->yn_used, ct, remain);

        ct_len -= remain;
        pt += remain;
        ct += remain;
        ctx->yn_used = (ctx->yn_used + remain) & 0xf;

        if (!ctx->yn_used)
            _gcm_ghash_4(ctx->tbl, ctx->yn, 16, (const unsigned char(*)[16])ctx->h);

        if (!ct_len)
            return;
    }

    i = ct_len & 0xfffffff0;

    ctr_dec(pt, ct, i, ctx->ctr, &ctx->key);

    if (i)
        _gcm_ghash_4(ctx->tbl, ct, i, (const unsigned char(*)[16])ctx->h);

    ct_len &= 0xf;

    if (!ct_len)
        return;

    pt += i;
    ct += i;

    lea_encrypt(ctx->yn, ctx->ctr, &ctx->key);
    ctr128_inc_aligned(ctx->ctr);
    ctx->yn_used = ct_len;

    for (ct_len--; ct_len >= 0; ct_len--) {
        pt[ct_len] = ctx->yn[ct_len] ^ ct[ct_len];
        ctx->yn[ct_len] = ct[ct_len];
    }
}
__host__ __device__ int gcm_final_4(LEA_GCM_CTX* ctx, unsigned char* tag, int tag_len)
{
    unsigned char tmp[16];

    if (!ctx || !tag) {
        memset(ctx, 0, sizeof(LEA_GCM_CTX));
        return -1;
    }
    if (tag_len < 4) {
        memset(ctx, 0, sizeof(LEA_GCM_CTX));
        return -1;
    }

    if (ctx->yn_used)
        _gcm_ghash_4(ctx->tbl, ctx->yn, ctx->yn_used, (const unsigned char(*)[16])ctx->h);

    memset(tmp, 0, 16);

    ctx->aad_len <<= 3;
    ctx->ct_len <<= 3;

    tmp[4] = ctx->aad_len >> 24;
    tmp[5] = ctx->aad_len >> 16;
    tmp[6] = ctx->aad_len >> 8;
    tmp[7] = ctx->aad_len;

    tmp[12] = ctx->ct_len >> 24;
    tmp[13] = ctx->ct_len >> 16;
    tmp[14] = ctx->ct_len >> 8;
    tmp[15] = ctx->ct_len;

    _gcm_ghash_4(ctx->tbl, tmp, 16, (const unsigned char(*)[16])ctx->h);

    XOR8x16(ctx->tbl, ctx->tbl, ctx->ek0);

    if (ctx->is_encrypt)
        memcpy(tag, ctx->tbl, tag_len);
    else
    {
        for (tag_len--; tag_len >= 0; tag_len--)
        {
            if (ctx->tbl[tag_len] != tag[tag_len]) {
                memset(ctx->ctr, 0, 16);
                memset(ctx->ek0, 0, 16);
                memset(ctx->tbl, 0, 16);
                memset(ctx->yn, 0, 16);
                ctx->yn_used = 0;

                return -1;

            }

        }
    }
    ctx->ct_len = 0;

    memset(ctx->ctr, 0, 16);
    memset(ctx->ek0, 0, 16);
    memset(ctx->tbl, 0, 16);
    memset(ctx->yn, 0, 16);
    ctx->yn_used = 0;

    return 0;
}
//8bit table version GCM
__host__ __device__ void gcm_set_ctr_8(LEA_GCM_CTX* ctx, const unsigned char* iv, int iv_len)
{
    int tmp_iv_len = iv_len;

    if (!ctx || !iv) {
        return;
    }
    if (iv_len < 0) {
        return;
    }

    ctx->ct_len = 0;

    if (iv_len == 12)
    {
        memcpy(ctx->ctr, iv, 12);
        ctx->ctr[15] = 1;
    }
    else
    {
        for (; iv_len >= 16; iv_len -= 16, iv += 16)
            _gcm_ghash_8(ctx->ctr, iv, 16, (const unsigned char(*)[16])ctx->h);

        if (iv_len)
            _gcm_ghash_8(ctx->ctr, iv, iv_len, (const unsigned char(*)[16])ctx->h);

        tmp_iv_len <<= 3;
        ctx->yn[12] = (tmp_iv_len >> 24) & 0xff;
        ctx->yn[13] = (tmp_iv_len >> 16) & 0xff;
        ctx->yn[14] = (tmp_iv_len >> 8) & 0xff;
        ctx->yn[15] = (tmp_iv_len) & 0xff;
        _gcm_ghash_8(ctx->ctr, ctx->yn, 16, (const unsigned char(*)[16])ctx->h);
        memset(ctx->yn, 0, 16);
    }

    lea_encrypt(ctx->ek0, ctx->ctr, &ctx->key);

    ctr128_inc_aligned(ctx->ctr);
}
__host__ __device__ void gcm_set_aad_8(LEA_GCM_CTX* ctx, const unsigned char* aad, int aad_len)
{
    if (!ctx) {
        return;
    }
    if (aad_len <= 0) {
        return;
    }
    if (!aad) {
        return;
    }
    ctx->aad_len = aad_len;

    _gcm_ghash_8(ctx->tbl, aad, aad_len, (const unsigned char(*)[16])ctx->h);
}
__host__ __device__ void gcm_enc_8(LEA_GCM_CTX* ctx, unsigned char* ct, const unsigned char* pt, int pt_len) {
    int remain, i;

    if (!ctx || !ct || !pt) {
        return;
    }
    if (pt_len < 0) {
        return;
    }

    ctx->is_encrypt = 1;
    ctx->ct_len += pt_len;

    if (!pt_len) {
        printf("\nreturn\n");
        return;
    }

    if (ctx->yn_used)
    {
        remain = 16 - ctx->yn_used;

        if (remain > pt_len)
            remain = pt_len;

        for (i = 0; i < remain; i++)
            ctx->yn[ctx->yn_used + i] ^= pt[i];

        memcpy(ct, ctx->yn + ctx->yn_used, remain);

        pt_len -= remain;
        pt += remain;
        ct += remain;
        ctx->yn_used = (ctx->yn_used + remain) & 0xf;

        if (!ctx->yn_used)
            _gcm_ghash_8(ctx->tbl, ctx->yn, 16, (const unsigned char(*)[16])ctx->h);

        if (!pt_len)
            return;
    }

    i = pt_len & 0xfffffff0;

    ctr_enc(ct, pt, i, ctx->ctr, &ctx->key);

    if (i)
        _gcm_ghash_8(ctx->tbl, ct, i, (const unsigned char(*)[16])ctx->h);

    pt_len &= 0xf;

    if (!pt_len)
        return;

    pt += i;
    ct += i;

    lea_encrypt(ctx->yn, ctx->ctr, &ctx->key);
    ctr128_inc_aligned(ctx->ctr);
    ctx->yn_used = pt_len;

    for (pt_len--; pt_len >= 0; pt_len--)
        ct[pt_len] = ctx->yn[pt_len] = ctx->yn[pt_len] ^ pt[pt_len];
}
__host__ __device__ void gcm_dec_8(LEA_GCM_CTX* ctx, unsigned char* pt, const unsigned char* ct, int ct_len)
{
    int remain, i;

    if (!ctx || !pt || !ct) {
        return;
    }
    if (ct_len < 0) {
        return;
    }

    ctx->is_encrypt = 0;
    ctx->ct_len += ct_len;

    if (!ct_len)
        return;

    if (ctx->yn_used)
    {
        remain = 16 - ctx->yn_used;

        if (remain > ct_len)
            remain = ct_len;

        for (i = 0; i < remain; i++)
            pt[i] = ctx->yn[ctx->yn_used + i] ^ ct[i];

        memcpy(ctx->yn + ctx->yn_used, ct, remain);

        ct_len -= remain;
        pt += remain;
        ct += remain;
        ctx->yn_used = (ctx->yn_used + remain) & 0xf;

        if (!ctx->yn_used)
            _gcm_ghash_8(ctx->tbl, ctx->yn, 16, (const unsigned char(*)[16])ctx->h);

        if (!ct_len)
            return;
    }

    i = ct_len & 0xfffffff0;

    ctr_dec(pt, ct, i, ctx->ctr, &ctx->key);

    if (i)
        _gcm_ghash_8(ctx->tbl, ct, i, (const unsigned char(*)[16])ctx->h);

    ct_len &= 0xf;

    if (!ct_len)
        return;

    pt += i;
    ct += i;

    lea_encrypt(ctx->yn, ctx->ctr, &ctx->key);
    ctr128_inc_aligned(ctx->ctr);
    ctx->yn_used = ct_len;

    for (ct_len--; ct_len >= 0; ct_len--) {
        pt[ct_len] = ctx->yn[ct_len] ^ ct[ct_len];
        ctx->yn[ct_len] = ct[ct_len];
    }
}
__host__ __device__ int gcm_final_8(LEA_GCM_CTX* ctx, unsigned char* tag, int tag_len)
{
    unsigned char tmp[16];

    if (!ctx || !tag) {
        memset(ctx, 0, sizeof(LEA_GCM_CTX));
        return -1;
    }
    if (tag_len < 4) {
        memset(ctx, 0, sizeof(LEA_GCM_CTX));
        return -1;
    }

    if (ctx->yn_used)
        _gcm_ghash_8(ctx->tbl, ctx->yn, ctx->yn_used, (const unsigned char(*)[16])ctx->h);

    memset(tmp, 0, 16);

    ctx->aad_len <<= 3;
    ctx->ct_len <<= 3;

    tmp[4] = ctx->aad_len >> 24;
    tmp[5] = ctx->aad_len >> 16;
    tmp[6] = ctx->aad_len >> 8;
    tmp[7] = ctx->aad_len;

    tmp[12] = ctx->ct_len >> 24;
    tmp[13] = ctx->ct_len >> 16;
    tmp[14] = ctx->ct_len >> 8;
    tmp[15] = ctx->ct_len;

    _gcm_ghash_8(ctx->tbl, tmp, 16, (const unsigned char(*)[16])ctx->h);

    XOR8x16(ctx->tbl, ctx->tbl, ctx->ek0);

    if (ctx->is_encrypt) {
        memcpy(tag, ctx->tbl, tag_len);
    }

    else
    {
        for (tag_len--; tag_len >= 0; tag_len--)
        {
            if (ctx->tbl[tag_len] != tag[tag_len]) {
                memset(ctx->ctr, 0, 16);
                memset(ctx->ek0, 0, 16);
                memset(ctx->tbl, 0, 16);
                memset(ctx->yn, 0, 16);
                ctx->yn_used = 0;

                return -1;

            }

        }
    }
    ctx->ct_len = 0;

    memset(ctx->ctr, 0, 16);
    memset(ctx->ek0, 0, 16);
    memset(ctx->tbl, 0, 16);
    memset(ctx->yn, 0, 16);
    ctx->yn_used = 0;

    return 0;
}



__global__ void parallel_enc_GHASH(uint8_t* tag, uint8_t* aad, uint8_t* ctr, uint8_t* pt, uint8_t* H, LEA_KEY* key, uint8_t* Y) {
    /*
    iv[12]
    pt[128]
    ctr[16 * 7] //ctr0 = Y로 바꿀것 , ctr1,...,ctr7-> 카운터들   인덱스 조절이 필요함
    aad[16 * 16]//앞은 aad[8][16] / enc(counter) // len으로 채울 것임
    mk[16]
    tag[16]
    */

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    //병렬화 암호화 코드
    lea_encrypt(aad + (tid / 8) * 16 * 16 + 7 * 16 + (tid % 8) * 16, ctr + tid * 16, key);
    XOR8x16(aad + (tid / 8) * 16 * 16 + 7 * 16 + (tid % 8) * 16, pt + tid * 16, aad + (tid / 8) * 16 * 16 + 7 * 16 + (tid % 8) * 16);

    //병렬화 GHASH코드
    parallel_ghash1(ctr + tid * 16, aad + (tid / 8) * 16 * 16 + (tid % 8) * 16, aad + (tid / 8) * 16 * 16 + (tid % 8) * 16 + 8 * 16, H + 3 * 16);
    parallel_ghash2(aad + tid * 16, ctr + (tid / 4) * 8 * 16 + (tid % 4) * 16, ctr + (tid / 4) * 8 * 16 + (tid % 4) * 16 + 4 * 16, H + 2 * 16);
    parallel_ghash3(ctr + tid * 16, aad + (tid / 2) * 4 * 16 + (tid % 2) * 16, aad + (tid / 2) * 4 * 16 + (tid % 2) * 16 + 2 * 16, H + 1 * 16);
    parallel_ghash_last(aad + tid * 16, ctr + tid * 2 * 16, ctr + tid * 2 * 16 + 16, H + 1 * 16, H);

    //tag XOR
    XOR8x16(tag + tid * 16, Y, aad + tid * 16);

}

//#define Alg_num 4           //1 block 8 thread = 1 alg, 1 block 32 thread = 4 alg
void test_GPU_parallel(int BlockSize, int ThreadSize) {//이게 약간 main문 같은것 다 박으면 됨
    int mk_len = 16; //klen
    int iv_len = 12; //nlen
    int pt_len = 16 * 8;	//plen
    int aad_len = 112;//alen
    int tag_len = 16;//tlen
    int ct_len = pt_len;

    int Alg_num = ThreadSize / 4;

    uint8_t iv[12] = {
        0x67,0x61,0x32,0x6D,0xE2,0x80,0x63,0x78,0x2E,0x96,0x72,0x98
    }; // 넘기는 값이 아님->Counter 값 사전 계산을 위함
    uint8_t cpu_pt[16 * 8] = {
        0x29,0xDA,0x4D,0x54,0x4C,0xAC,0x60,0xB8,0x83,0x1E,0x0A,0xFB,0x3B,0xB4,0x4E,0x5F,
        0xBC,0x68,0xCC,0x59,0xB9,0xF1,0xEF,0xF2,0x25,0x45,0x67,0x6D,0x49,0x5D,0xDA,0x2A,
        0x14,0x38,0xD6,0xCA,0xB2,0x22,0x0B,0x94,0x60,0x36,0xB7,0x17,0x7E,0x22,0x61,0x11,
        0xE5,0x2A,0xCA,0x90,0x7C,0x70,0x21,0x57,0x06,0x72,0x76,0x83,0x3E,0xD4,0x71,0x6F,
        0x26,0x60,0x44,0xD4,0x9C,0x4B,0xDE,0x35,0x2E,0xB9,0x61,0x7A,0x2F,0x84,0xD7,0xDB,
        0x0A,0x39,0x21,0xFF,0xD7,0x64,0x2F,0x65,0x2C,0x0E,0x77,0x04,0x36,0x83,0x9F,0x2E,
        0x08,0x59,0x7D,0xBA,0x32,0xAD,0x42,0x62,0x96,0xB0,0xF2,0x6A,0x77,0x63,0x18,0x83,
        0xD3,0x9E,0xB7,0xEE,0xF3,0x57,0x83,0xE4,0x40,0x21,0x51,0x36,0x58,0xF3,0xCD,0x31

    };
    uint8_t* pt = NULL;
    pt = (uint8_t*)malloc(sizeof(uint8_t) * BlockSize * Alg_num * 16 * 8);
    for (int i = 0; i < BlockSize * Alg_num; i++) {
        memcpy(pt + 16 * 8 * i, cpu_pt, sizeof(uint8_t) * 16 * 8);
    }
    uint8_t cpu_aad[16 * 16] = {  //앞은 aad / enc(counter) // len으로 채울 것임
        0xE7,0x60,0x76,0xA5,0xF7,0xD0,0x84,0x9C,0xE9,0xB4,0xEE,0x62,0xCD,0xAB,0x61,0x3E,
        0xDA,0xDF,0xF5,0x3A,0x05,0x42,0x5E,0x84,0xD3,0x17,0x66,0x14,0x6B,0xB5,0x9B,0x8F,
        0xEF,0x87,0x44,0x6B,0x0C,0x58,0x9A,0x55,0xD0,0xCD,0x35,0xCF,0xBC,0x33,0xC5,0x1E,
        0x1B,0x7B,0x02,0x63,0x92,0x83,0xE6,0x08,0xF7,0x98,0x19,0x5D,0x66,0x38,0xAD,0x83,
        0x92,0x1C,0x0B,0x8A,0x5A,0x76,0x2D,0xD6,0x9D,0x77,0x0E,0xC7,0x4E,0x14,0x5F,0x99,
        0xCB,0x0A,0x7C,0x88,0x6A,0x1F,0x9E,0x38,0x55,0xAB,0x52,0x55,0x99,0x58,0x2A,0x76,
        0x79,0x26,0xB2,0x48,0x08,0xB2,0xA5,0xE8,0xF7,0xB7,0x61,0x90,0xAB,0xD5,0x99,0x76,
        0x00,
    };
    uint8_t* aad = NULL;
    aad = (uint8_t*)malloc(sizeof(uint8_t) * BlockSize * Alg_num * 16 * 16);

    // len값 저장
    aad_len <<= 3;
    ct_len <<= 3;
    cpu_aad[16 * 15 + 4] = aad_len >> 24;
    cpu_aad[16 * 15 + 5] = aad_len >> 16;
    cpu_aad[16 * 15 + 6] = aad_len >> 8;
    cpu_aad[16 * 15 + 7] = aad_len;
    cpu_aad[16 * 15 + 12] = ct_len >> 24;
    cpu_aad[16 * 15 + 13] = ct_len >> 16;
    cpu_aad[16 * 15 + 14] = ct_len >> 8;
    cpu_aad[16 * 15 + 15] = ct_len;
    //printf("hi\n");
    for (int i = 0; i < BlockSize * Alg_num; i++) {
        memcpy(aad + 16 * 16 * i, cpu_aad, sizeof(uint8_t) * 16 * 16);
        //print_hex(aad + 16*16 * i, 16*16);
    }
    uint8_t mk[16] = {
        0x43,0x60,0x77,0xD9,0xEF,0x6A,0x74,0xDC,0x3F,0xB2,0x37,0xFC,0xE6,0xEB,0x3D,0x11
    };
    //lea_encrypt(h, zero, mk);
    uint8_t cpu_tag[16] = {
        0x00,
    };
    uint8_t* tag = NULL;
    tag = (uint8_t*)malloc(sizeof(uint8_t) * BlockSize * Alg_num * 16);
    for (int i = 0; i < BlockSize * Alg_num; i++) {
        memcpy(tag + 16 * i, cpu_tag, sizeof(uint8_t) * 16);
    }

    //key 만들기   
    LEA_KEY* key;
    key = (LEA_KEY*)malloc(sizeof(LEA_KEY) * BlockSize * Alg_num);
    memset(key, 0, sizeof(LEA_KEY) * BlockSize * Alg_num);
    lea_set_key(key, mk, mk_len);
    LEA_KEY* g_key = NULL;
    hipMalloc((void**)&g_key, sizeof(LEA_KEY));


    //H,Y제작
    uint8_t ZERO[16] = { 0x00, };
    uint8_t* H = NULL;
    H = (uint8_t*)malloc(sizeof(uint8_t) * 16 * 4);
    lea_encrypt(H, ZERO, key);
    gcm_gfmul_m(H + 16, H, H);  //--> ctx->sub_h[1] = H^2
    gcm_gfmul_m(H + 16 * 2, H + 16, H + 16);  //--> ctx->sub_h[2] = H^4
    gcm_gfmul_m(H + 16 * 3, H + 16 * 2, H + 16 * 2);  //--> ctx->sub_h[3] = H^8

    //Y제작
    uint8_t cpu_Y[16] = { 0x00, };
    uint8_t* Y = NULL;
    Y = (uint8_t*)malloc(sizeof(uint8_t) * BlockSize * Alg_num * 16);
    memcpy(cpu_Y, iv, 12);
    cpu_Y[15] = 1;
    lea_encrypt(cpu_Y, cpu_Y, key);
    for (int i = 0; i < BlockSize * Alg_num; i++) {
        memcpy(Y + 16 * i, cpu_Y, 16);
    }
    uint8_t* ctr = NULL;
    ctr = (uint8_t*)malloc(sizeof(uint8_t) * BlockSize * Alg_num * 16 * 8);
    memset(ctr, 0, sizeof(uint8_t) * BlockSize * Alg_num * 16 * 8);
    memcpy(&ctr[0], iv, 12);//CTR0값
    ctr[15] = 2;
    memcpy(&ctr[1 << 4], iv, 12);//CTR1값
    ctr[(1 << 4) + 15] = 3;
    memcpy(&ctr[2 << 4], iv, 12);//CTR2값
    ctr[(2 << 4) + 15] = 4;
    memcpy(&ctr[3 << 4], iv, 12);//CTR3값
    ctr[(3 << 4) + 15] = 5;
    memcpy(&ctr[4 << 4], iv, 12);//CTR4값
    ctr[(4 << 4) + 15] = 6;
    memcpy(&ctr[5 << 4], iv, 12);//CTR5값
    ctr[(5 << 4) + 15] = 7;
    memcpy(&ctr[6 << 4], iv, 12);//CTR6값
    ctr[(6 << 4) + 15] = 8;
    memcpy(&ctr[7 << 4], iv, 12);//CTR6값
    ctr[(7 << 4) + 15] = 9;
    for (int i = 0; i < BlockSize * Alg_num; i++) {
        memcpy(ctr + 16 * 8 * i, ctr, sizeof(uint8_t) * 16 * 8);
    }
    //넘겨 줘야할 값들(g_tag, g_aad, g_ctr, g_pt, &g_key,g_H)
    //Y
    uint8_t* g_Y = NULL;
    hipMalloc((void**)&g_Y, BlockSize * Alg_num * 16 * sizeof(uint8_t));
    //H
    uint8_t* g_H = NULL;
    hipMalloc((void**)&g_H, BlockSize * Alg_num * 16 * 4 * sizeof(uint8_t));
    // pt
    uint8_t* g_pt = NULL;
    hipMalloc((void**)&g_pt, BlockSize * Alg_num * 16 * 8 * sizeof(uint8_t));
    // ctr -> 암호화 값 저장
    uint8_t* g_ctr = NULL;
    hipMalloc((void**)&g_ctr, BlockSize * Alg_num * 16 * 8 * sizeof(uint8_t));
    // aad //len값 마지막에 포함
    uint8_t* g_aad = NULL;
    hipMalloc((void**)&g_aad, BlockSize * Alg_num * 16 * 16 * sizeof(uint8_t));
    //tag 결과값 저장 
    uint8_t* g_tag = NULL;
    hipMalloc((void**)&g_tag, BlockSize * Alg_num * 16 * sizeof(uint8_t));



    //cuda에 값 복제
    hipMemcpy((void**)g_H, H, BlockSize * Alg_num * 16 * 4, hipMemcpyHostToDevice);
    hipMemcpy((void**)g_pt, pt, BlockSize * Alg_num * 16 * 8, hipMemcpyHostToDevice);
    hipMemcpy((void**)g_ctr, ctr, BlockSize * Alg_num * 16 * 8, hipMemcpyHostToDevice);
    hipMemcpy((void**)g_key, key, sizeof(LEA_KEY), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_tag, tag, BlockSize * Alg_num * 16 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_aad, aad, BlockSize * Alg_num * 16 * 16 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_Y, Y, BlockSize * Alg_num * 16 * sizeof(uint8_t), hipMemcpyHostToDevice);

    //성능측정
    hipEvent_t start, stop;
    float elapsed_time_ms = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    //Global
    parallel_enc_GHASH << < (BlockSize + 31) / 32, ThreadSize >> > (g_tag, g_aad, g_ctr, g_pt, g_H, g_key, g_Y);


    hipMemcpy(aad, g_aad, BlockSize * Alg_num * sizeof(uint8_t) * 16 * 16, hipMemcpyDeviceToHost);

    


    hipDeviceSynchronize();
    //성능측정
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    printf("my code\n");
    printf("elapsed_time_ms is %4.4f\n", elapsed_time_ms);
    printf("Performance : %4.2f GCM time per second \n", BlockSize * Alg_num / ((elapsed_time_ms / 1000)));
    //cuda->cpu
    hipMemcpy(tag, g_tag, BlockSize * Alg_num * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipFree(g_pt);
    hipFree(g_tag);
    hipFree(g_aad);
    hipFree(g_ctr);
    hipFree(g_H);
    hipFree(g_Y);
    hipFree(g_key);

    free(pt);
    free(ctr);
    free(aad);
    free(H);
    free(key);
    free(tag);
    free(Y);
}
__global__ void GCM_REF(uint8_t* mk, int mk_len, uint8_t* iv, int iv_len, uint8_t* aad, int aad_len, uint8_t* ct, uint8_t* pt, int pt_len, uint8_t* tag, int tag_len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    LEA_GCM_CTX ctx;
    _lea_gcm_init_m(&ctx, mk + tid * 16, mk_len);
    gcm_set_ctr(&ctx, iv + tid * 12, iv_len);
    gcm_set_aad(&ctx, aad + tid * 112, aad_len);
    gcm_enc(&ctx, ct + tid * 128, pt + tid * 128, pt_len);
    gcm_final(&ctx, tag + tid * 16, tag_len);
}
void test_GPU_REF(int BlockSize, int ThreadSize) {//이게 약간 main문 같은것 다 박으면 됨
    //ctx
   // LEA_GCM_CTX* ctx;
    //ctx = (LEA_GCM_CTX*)malloc(BlockSize * ThreadSize * sizeof(LEA_GCM_CTX));
    //iv
    uint8_t cpu_iv[12] = {
        0x67,0x61,0x32,0x6D,0xE2,0x80,0x63,0x78,0x2E,0x96,0x72,0x98
    };
    uint8_t* iv = NULL;
    iv = (uint8_t*)malloc(BlockSize * ThreadSize * 12 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(iv + 12 * i, cpu_iv, 12 * sizeof(uint8_t));
    }
    //pt
    uint8_t cpu_pt[128] = {
        0x29,0xDA,0x4D,0x54,0x4C,0xAC,0x60,0xB8,0x83,0x1E,0x0A,0xFB,0x3B,0xB4,0x4E,0x5F,
        0xBC,0x68,0xCC,0x59,0xB9,0xF1,0xEF,0xF2,0x25,0x45,0x67,0x6D,0x49,0x5D,0xDA,0x2A,
        0x14,0x38,0xD6,0xCA,0xB2,0x22,0x0B,0x94,0x60,0x36,0xB7,0x17,0x7E,0x22,0x61,0x11,
        0xE5,0x2A,0xCA,0x90,0x7C,0x70,0x21,0x57,0x06,0x72,0x76,0x83,0x3E,0xD4,0x71,0x6F,
        0x26,0x60,0x44,0xD4,0x9C,0x4B,0xDE,0x35,0x2E,0xB9,0x61,0x7A,0x2F,0x84,0xD7,0xDB,
        0x0A,0x39,0x21,0xFF,0xD7,0x64,0x2F,0x65,0x2C,0x0E,0x77,0x04,0x36,0x83,0x9F,0x2E,
        0x08,0x59,0x7D,0xBA,0x32,0xAD,0x42,0x62,0x96,0xB0,0xF2,0x6A,0x77,0x63,0x18,0x83,
        0xD3,0x9E,0xB7,0xEE,0xF3,0x57,0x83,0xE4,0x40,0x21,0x51,0x36,0x58,0xF3,0xCD,0x31
    };
    uint8_t* pt = NULL;
    pt = (uint8_t*)malloc(BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(pt + 128 * i, cpu_pt, 128 * sizeof(uint8_t));
    }
    //ct
    uint8_t cpu_ct[128] = { 0x00, };
    uint8_t* ct = NULL;
    ct = (uint8_t*)malloc(BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(ct + 128 * i, cpu_ct, 128 * sizeof(uint8_t));
    }
    //aad
    uint8_t cpu_aad[112] = {
        0xE7,0x60,0x76,0xA5,0xF7,0xD0,0x84,0x9C,0xE9,0xB4,0xEE,0x62,0xCD,0xAB,0x61,0x3E,
        0xDA,0xDF,0xF5,0x3A,0x05,0x42,0x5E,0x84,0xD3,0x17,0x66,0x14,0x6B,0xB5,0x9B,0x8F,
        0xEF,0x87,0x44,0x6B,0x0C,0x58,0x9A,0x55,0xD0,0xCD,0x35,0xCF,0xBC,0x33,0xC5,0x1E,
        0x1B,0x7B,0x02,0x63,0x92,0x83,0xE6,0x08,0xF7,0x98,0x19,0x5D,0x66,0x38,0xAD,0x83,
        0x92,0x1C,0x0B,0x8A,0x5A,0x76,0x2D,0xD6,0x9D,0x77,0x0E,0xC7,0x4E,0x14,0x5F,0x99,
        0xCB,0x0A,0x7C,0x88,0x6A,0x1F,0x9E,0x38,0x55,0xAB,0x52,0x55,0x99,0x58,0x2A,0x76,
        0x79,0x26,0xB2,0x48,0x08,0xB2,0xA5,0xE8,0xF7,0xB7,0x61,0x90,0xAB,0xD5,0x99,0x76
    };
    uint8_t* aad = NULL;
    aad = (uint8_t*)malloc(BlockSize * ThreadSize * 112 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(aad + 112 * i, cpu_aad, 112 * sizeof(uint8_t));
    }
    //mk
    uint8_t cpu_mk[16] = {
        0x43,0x60,0x77,0xD9,0xEF,0x6A,0x74,0xDC,0x3F,0xB2,0x37,0xFC,0xE6,0xEB,0x3D,0x11
    };
    uint8_t* mk = NULL;
    mk = (uint8_t*)malloc(BlockSize * ThreadSize * 16 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(mk + 16 * i, cpu_mk, 16 * sizeof(uint8_t));
    }
    //tag
    uint8_t cpu_tag[16] = {
        0x00,
    };
    uint8_t* tag = NULL;
    tag = (uint8_t*)malloc(BlockSize * ThreadSize * 16 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(tag + 16 * i, cpu_tag, 16 * sizeof(uint8_t));
    }

    int mk_len = 16; //klen
    int iv_len = 12; //nlen
    int pt_len = 16 * 8;	//plen
    int aad_len = 112;//alen
    int tag_len = 16;//tlen
    int ct_len = pt_len;
    //넘겨 줘야할 값들(g_tag, g_aad, g_ctr, g_pt, &g_key,g_H)
    //LEA_GCM_CTX* g_ctx = NULL;
    //hipMalloc((void**)g_ctx, BlockSize * ThreadSize * sizeof(LEA_GCM_CTX));
    // pt
    uint8_t* g_pt = NULL;
    hipMalloc((void**)&g_pt, BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    // ct
    uint8_t* g_ct = NULL;
    hipMalloc((void**)&g_ct, BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    // aad //len값 마지막에 포함
    uint8_t* g_aad = NULL;
    hipMalloc((void**)&g_aad, BlockSize * ThreadSize * 112 * sizeof(uint8_t));
    //tag 결과값 저장 
    uint8_t* g_tag = NULL;
    hipMalloc((void**)&g_tag, BlockSize * ThreadSize * 16 * sizeof(uint8_t));
    //iv  
    uint8_t* g_iv = NULL;
    hipMalloc((void**)&g_iv, BlockSize * ThreadSize * 12 * sizeof(uint8_t));
    //mk
    uint8_t* g_mk = NULL;
    hipMalloc((void**)&g_mk, BlockSize * ThreadSize * 16 * sizeof(uint8_t));


    //cuda에 값 복제
    hipMemcpy((void**)g_pt, pt, BlockSize * ThreadSize * 128 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_mk, mk, BlockSize * ThreadSize * 16 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_tag, tag, BlockSize * ThreadSize * 16 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_aad, aad, BlockSize * ThreadSize * 112 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_iv, iv, BlockSize * ThreadSize * 12 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_ct, ct, BlockSize * ThreadSize * 128 * sizeof(uint8_t), hipMemcpyHostToDevice);

    // LEA_GCM_CTX* ctx = NULL;
    // ctx = (LEA_GCM_CTX*)malloc(BlockSize * ThreadSize * sizeof(LEA_GCM_CTX));
    // memset(ctx, 0, sizeof(LEA_GCM_CTX) * ThreadSize * BlockSize);
    // LEA_GCM_CTX* g_ctx = NULL;
    // hipMalloc((void**)g_ctx, BlockSize * ThreadSize * sizeof(LEA_GCM_CTX));
    // hipMemcpy((void**)g_ctx, ctx, BlockSize* ThreadSize * sizeof(LEA_GCM_CTX), hipMemcpyHostToDevice);


     //성능측정
    hipEvent_t start, stop;
    float elapsed_time_ms = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Global
    GCM_REF << < BlockSize, ThreadSize >> > (g_mk, mk_len, g_iv, iv_len, g_aad, aad_len, g_ct, g_pt, pt_len, g_tag, tag_len);

    hipDeviceSynchronize();
    //성능측정
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    printf("reference code\n");
    printf("elapsed_time_ms is %4.4f\n", elapsed_time_ms);
    printf("Performance : %4.2f GCM time per second \n", BlockSize * ThreadSize / ((elapsed_time_ms / 1000)));

    //cuda->cpu
    hipMemcpy(tag, g_tag, BlockSize * 16 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(aad, g_aad, BlockSize * 112 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(ct, g_ct, BlockSize * 128 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(pt, g_pt, BlockSize * 128 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    //hipMemcpy(ctx, g_ctx, BlockSize  * ThreadSize * sizeof(LEA_GCM_CTX), hipMemcpyDeviceToHost);



    hipFree(g_pt);
    hipFree(g_tag);
    hipFree(g_aad);
    hipFree(g_mk);
    hipFree(g_ct);
    //hipFree(g_ctx);
    hipFree(g_iv);

    free(pt);
    //free(ctx);
    free(aad);
    free(mk);
    free(iv);
    free(tag);
    free(ct);
}
__global__ void GCM_4bit_table(uint8_t* mk, int mk_len, uint8_t* iv, int iv_len, uint8_t* aad, int aad_len, uint8_t* ct, uint8_t* pt, int pt_len, uint8_t* tag, int tag_len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    LEA_GCM_CTX ctx;
    _lea_gcm_init_4(&ctx, mk + tid * 16, mk_len);
    gcm_set_ctr_4(&ctx, iv + tid * 12, iv_len);
    gcm_set_aad_4(&ctx, aad + tid * 112, aad_len);
    gcm_enc_4(&ctx, ct + tid * 128, pt + tid * 128, pt_len);
    gcm_final_4(&ctx, tag + tid * 16, tag_len);

}
void test_GPU_4bit_table(int BlockSize, int ThreadSize) {//이게 약간 main문 같은것 다 박으면 됨
    //ctx
   // LEA_GCM_CTX* ctx;
    //ctx = (LEA_GCM_CTX*)malloc(BlockSize * ThreadSize * sizeof(LEA_GCM_CTX));
    //iv
    uint8_t cpu_iv[12] = {
        0x67,0x61,0x32,0x6D,0xE2,0x80,0x63,0x78,0x2E,0x96,0x72,0x98
    };
    uint8_t* iv = NULL;
    iv = (uint8_t*)malloc(BlockSize * ThreadSize * 12 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(iv + 12 * i, cpu_iv, 12 * sizeof(uint8_t));
    }
    //pt
    uint8_t cpu_pt[128] = {
        0x29,0xDA,0x4D,0x54,0x4C,0xAC,0x60,0xB8,0x83,0x1E,0x0A,0xFB,0x3B,0xB4,0x4E,0x5F,
        0xBC,0x68,0xCC,0x59,0xB9,0xF1,0xEF,0xF2,0x25,0x45,0x67,0x6D,0x49,0x5D,0xDA,0x2A,
        0x14,0x38,0xD6,0xCA,0xB2,0x22,0x0B,0x94,0x60,0x36,0xB7,0x17,0x7E,0x22,0x61,0x11,
        0xE5,0x2A,0xCA,0x90,0x7C,0x70,0x21,0x57,0x06,0x72,0x76,0x83,0x3E,0xD4,0x71,0x6F,
        0x26,0x60,0x44,0xD4,0x9C,0x4B,0xDE,0x35,0x2E,0xB9,0x61,0x7A,0x2F,0x84,0xD7,0xDB,
        0x0A,0x39,0x21,0xFF,0xD7,0x64,0x2F,0x65,0x2C,0x0E,0x77,0x04,0x36,0x83,0x9F,0x2E,
        0x08,0x59,0x7D,0xBA,0x32,0xAD,0x42,0x62,0x96,0xB0,0xF2,0x6A,0x77,0x63,0x18,0x83,
        0xD3,0x9E,0xB7,0xEE,0xF3,0x57,0x83,0xE4,0x40,0x21,0x51,0x36,0x58,0xF3,0xCD,0x31
    };
    uint8_t* pt = NULL;
    pt = (uint8_t*)malloc(BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(pt + 128 * i, cpu_pt, 128 * sizeof(uint8_t));
    }
    //ct
    uint8_t cpu_ct[128] = { 0x00, };
    uint8_t* ct = NULL;
    ct = (uint8_t*)malloc(BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(ct + 128 * i, cpu_ct, 128 * sizeof(uint8_t));
    }
    //aad
    uint8_t cpu_aad[112] = {
        0xE7,0x60,0x76,0xA5,0xF7,0xD0,0x84,0x9C,0xE9,0xB4,0xEE,0x62,0xCD,0xAB,0x61,0x3E,
        0xDA,0xDF,0xF5,0x3A,0x05,0x42,0x5E,0x84,0xD3,0x17,0x66,0x14,0x6B,0xB5,0x9B,0x8F,
        0xEF,0x87,0x44,0x6B,0x0C,0x58,0x9A,0x55,0xD0,0xCD,0x35,0xCF,0xBC,0x33,0xC5,0x1E,
        0x1B,0x7B,0x02,0x63,0x92,0x83,0xE6,0x08,0xF7,0x98,0x19,0x5D,0x66,0x38,0xAD,0x83,
        0x92,0x1C,0x0B,0x8A,0x5A,0x76,0x2D,0xD6,0x9D,0x77,0x0E,0xC7,0x4E,0x14,0x5F,0x99,
        0xCB,0x0A,0x7C,0x88,0x6A,0x1F,0x9E,0x38,0x55,0xAB,0x52,0x55,0x99,0x58,0x2A,0x76,
        0x79,0x26,0xB2,0x48,0x08,0xB2,0xA5,0xE8,0xF7,0xB7,0x61,0x90,0xAB,0xD5,0x99,0x76
    };
    uint8_t* aad = NULL;
    aad = (uint8_t*)malloc(BlockSize * ThreadSize * 112 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(aad + 112 * i, cpu_aad, 112 * sizeof(uint8_t));
    }
    //mk
    uint8_t cpu_mk[16] = {
        0x43,0x60,0x77,0xD9,0xEF,0x6A,0x74,0xDC,0x3F,0xB2,0x37,0xFC,0xE6,0xEB,0x3D,0x11
    };
    uint8_t* mk = NULL;
    mk = (uint8_t*)malloc(BlockSize * ThreadSize * 16 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(mk + 16 * i, cpu_mk, 16 * sizeof(uint8_t));
    }
    //tag
    uint8_t cpu_tag[16] = {
        0x00,
    };
    uint8_t* tag = NULL;
    tag = (uint8_t*)malloc(BlockSize * ThreadSize * 16 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(tag + 16 * i, cpu_tag, 16 * sizeof(uint8_t));
    }

    int mk_len = 16; //klen
    int iv_len = 12; //nlen
    int pt_len = 16 * 8;	//plen
    int aad_len = 112;//alen
    int tag_len = 16;//tlen
    int ct_len = pt_len;
    //넘겨 줘야할 값들(g_tag, g_aad, g_ctr, g_pt, &g_key,g_H)
    //LEA_GCM_CTX* g_ctx = NULL;
    //hipMalloc((void**)g_ctx, BlockSize * ThreadSize * sizeof(LEA_GCM_CTX));
    // pt
    uint8_t* g_pt = NULL;
    hipMalloc((void**)&g_pt, BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    // ct
    uint8_t* g_ct = NULL;
    hipMalloc((void**)&g_ct, BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    // aad //len값 마지막에 포함
    uint8_t* g_aad = NULL;
    hipMalloc((void**)&g_aad, BlockSize * ThreadSize * 112 * sizeof(uint8_t));
    //tag 결과값 저장 
    uint8_t* g_tag = NULL;
    hipMalloc((void**)&g_tag, BlockSize * ThreadSize * 16 * sizeof(uint8_t));
    //iv  
    uint8_t* g_iv = NULL;
    hipMalloc((void**)&g_iv, BlockSize * ThreadSize * 12 * sizeof(uint8_t));
    //mk
    uint8_t* g_mk = NULL;
    hipMalloc((void**)&g_mk, BlockSize * ThreadSize * 16 * sizeof(uint8_t));


    //cuda에 값 복제
    hipMemcpy((void**)g_pt, pt, BlockSize * ThreadSize * 128 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_mk, mk, BlockSize * ThreadSize * 16 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_tag, tag, BlockSize * ThreadSize * 16 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_aad, aad, BlockSize * ThreadSize * 112 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_iv, iv, BlockSize * ThreadSize * 12 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_ct, ct, BlockSize * ThreadSize * 128 * sizeof(uint8_t), hipMemcpyHostToDevice);
    //hipMemcpy((void**)g_ctx, ctx, BlockSize* ThreadSize * sizeof(LEA_GCM_CTX), hipMemcpyHostToDevice);


    //성능측정
    hipEvent_t start, stop;
    float elapsed_time_ms = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Global
    GCM_4bit_table << < BlockSize, ThreadSize >> > (g_mk, mk_len, g_iv, iv_len, g_aad, aad_len, g_ct, g_pt, pt_len, g_tag, tag_len);

    hipDeviceSynchronize();
    //성능측정
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    printf("4bit table version code\n");
    printf("elapsed_time_ms is %4.4f\n", elapsed_time_ms);
    printf("Performance : %4.2f GCM time per second \n", BlockSize * ThreadSize / ((elapsed_time_ms / 1000)));

    //cuda->cpu
    hipMemcpy(tag, g_tag, BlockSize * 16 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(aad, g_aad, BlockSize * 112 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(ct, g_ct, BlockSize * 128 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(pt, g_pt, BlockSize * 128 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    //hipMemcpy(ctx, g_ctx, BlockSize  * ThreadSize * sizeof(LEA_GCM_CTX), hipMemcpyDeviceToHost);



    hipFree(g_pt);
    hipFree(g_tag);
    hipFree(g_aad);
    hipFree(g_mk);
    hipFree(g_ct);
    //hipFree(g_ctx);
    hipFree(g_iv);

    free(pt);
    //free(ctx);
    free(aad);
    free(mk);
    free(iv);
    free(tag);
    free(ct);
}
__global__ void GCM_8bit_table(uint8_t* mk, int mk_len, uint8_t* iv, int iv_len, uint8_t* aad, int aad_len, uint8_t* ct, uint8_t* pt, int pt_len, uint8_t* tag, int tag_len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    LEA_GCM_CTX ctx;
    _lea_gcm_init_8(&ctx, mk + tid * 16, mk_len);
    gcm_set_ctr_8(&ctx, iv + tid * 12, iv_len);
    gcm_set_aad_8(&ctx, aad + tid * 112, aad_len);
    gcm_enc_8(&ctx, ct + tid * 128, pt + tid * 128, pt_len);
    gcm_final_8(&ctx, tag + tid * 16, tag_len);
}
void test_GPU_8bit_table(int BlockSize, int ThreadSize) {//이게 약간 main문 같은것 다 박으면 됨
    //ctx
   // LEA_GCM_CTX* ctx;
    //ctx = (LEA_GCM_CTX*)malloc(BlockSize * ThreadSize * sizeof(LEA_GCM_CTX));
    //iv
    uint8_t cpu_iv[12] = {
        0x67,0x61,0x32,0x6D,0xE2,0x80,0x63,0x78,0x2E,0x96,0x72,0x98
    };
    uint8_t* iv = NULL;
    iv = (uint8_t*)malloc(BlockSize * ThreadSize * 12 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(iv + 12 * i, cpu_iv, 12 * sizeof(uint8_t));
    }
    //pt
    uint8_t cpu_pt[128] = {
        0x29,0xDA,0x4D,0x54,0x4C,0xAC,0x60,0xB8,0x83,0x1E,0x0A,0xFB,0x3B,0xB4,0x4E,0x5F,
        0xBC,0x68,0xCC,0x59,0xB9,0xF1,0xEF,0xF2,0x25,0x45,0x67,0x6D,0x49,0x5D,0xDA,0x2A,
        0x14,0x38,0xD6,0xCA,0xB2,0x22,0x0B,0x94,0x60,0x36,0xB7,0x17,0x7E,0x22,0x61,0x11,
        0xE5,0x2A,0xCA,0x90,0x7C,0x70,0x21,0x57,0x06,0x72,0x76,0x83,0x3E,0xD4,0x71,0x6F,
        0x26,0x60,0x44,0xD4,0x9C,0x4B,0xDE,0x35,0x2E,0xB9,0x61,0x7A,0x2F,0x84,0xD7,0xDB,
        0x0A,0x39,0x21,0xFF,0xD7,0x64,0x2F,0x65,0x2C,0x0E,0x77,0x04,0x36,0x83,0x9F,0x2E,
        0x08,0x59,0x7D,0xBA,0x32,0xAD,0x42,0x62,0x96,0xB0,0xF2,0x6A,0x77,0x63,0x18,0x83,
        0xD3,0x9E,0xB7,0xEE,0xF3,0x57,0x83,0xE4,0x40,0x21,0x51,0x36,0x58,0xF3,0xCD,0x31
    };
    uint8_t* pt = NULL;
    pt = (uint8_t*)malloc(BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(pt + 128 * i, cpu_pt, 128 * sizeof(uint8_t));
    }
    //ct
    uint8_t cpu_ct[128] = { 0x00, };
    uint8_t* ct = NULL;
    ct = (uint8_t*)malloc(BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(ct + 128 * i, cpu_ct, 128 * sizeof(uint8_t));
    }
    //aad
    uint8_t cpu_aad[112] = {
        0xE7,0x60,0x76,0xA5,0xF7,0xD0,0x84,0x9C,0xE9,0xB4,0xEE,0x62,0xCD,0xAB,0x61,0x3E,
        0xDA,0xDF,0xF5,0x3A,0x05,0x42,0x5E,0x84,0xD3,0x17,0x66,0x14,0x6B,0xB5,0x9B,0x8F,
        0xEF,0x87,0x44,0x6B,0x0C,0x58,0x9A,0x55,0xD0,0xCD,0x35,0xCF,0xBC,0x33,0xC5,0x1E,
        0x1B,0x7B,0x02,0x63,0x92,0x83,0xE6,0x08,0xF7,0x98,0x19,0x5D,0x66,0x38,0xAD,0x83,
        0x92,0x1C,0x0B,0x8A,0x5A,0x76,0x2D,0xD6,0x9D,0x77,0x0E,0xC7,0x4E,0x14,0x5F,0x99,
        0xCB,0x0A,0x7C,0x88,0x6A,0x1F,0x9E,0x38,0x55,0xAB,0x52,0x55,0x99,0x58,0x2A,0x76,
        0x79,0x26,0xB2,0x48,0x08,0xB2,0xA5,0xE8,0xF7,0xB7,0x61,0x90,0xAB,0xD5,0x99,0x76
    };
    uint8_t* aad = NULL;
    aad = (uint8_t*)malloc(BlockSize * ThreadSize * 112 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(aad + 112 * i, cpu_aad, 112 * sizeof(uint8_t));
    }
    //mk
    uint8_t cpu_mk[16] = {
        0x43,0x60,0x77,0xD9,0xEF,0x6A,0x74,0xDC,0x3F,0xB2,0x37,0xFC,0xE6,0xEB,0x3D,0x11
    };
    uint8_t* mk = NULL;
    mk = (uint8_t*)malloc(BlockSize * ThreadSize * 16 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(mk + 16 * i, cpu_mk, 16 * sizeof(uint8_t));
    }
    //tag
    uint8_t cpu_tag[16] = {
        0x00,
    };
    uint8_t* tag = NULL;
    tag = (uint8_t*)malloc(BlockSize * ThreadSize * 16 * sizeof(uint8_t));
    for (int i = 0; i < BlockSize * ThreadSize; i++) {
        memcpy(tag + 16 * i, cpu_tag, 16 * sizeof(uint8_t));
    }

    int mk_len = 16; //klen
    int iv_len = 12; //nlen
    int pt_len = 16 * 8;	//plen
    int aad_len = 112;//alen
    int tag_len = 16;//tlen
    int ct_len = pt_len;
    //넘겨 줘야할 값들(g_tag, g_aad, g_ctr, g_pt, &g_key,g_H)
    //LEA_GCM_CTX* g_ctx = NULL;
    //hipMalloc((void**)g_ctx, BlockSize * ThreadSize * sizeof(LEA_GCM_CTX));
    // pt
    uint8_t* g_pt = NULL;
    hipMalloc((void**)&g_pt, BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    // ct
    uint8_t* g_ct = NULL;
    hipMalloc((void**)&g_ct, BlockSize * ThreadSize * 128 * sizeof(uint8_t));
    // aad //len값 마지막에 포함
    uint8_t* g_aad = NULL;
    hipMalloc((void**)&g_aad, BlockSize * ThreadSize * 112 * sizeof(uint8_t));
    //tag 결과값 저장 
    uint8_t* g_tag = NULL;
    hipMalloc((void**)&g_tag, BlockSize * ThreadSize * 16 * sizeof(uint8_t));
    //iv  
    uint8_t* g_iv = NULL;
    hipMalloc((void**)&g_iv, BlockSize * ThreadSize * 12 * sizeof(uint8_t));
    //mk
    uint8_t* g_mk = NULL;
    hipMalloc((void**)&g_mk, BlockSize * ThreadSize * 16 * sizeof(uint8_t));


    //cuda에 값 복제
    hipMemcpy((void**)g_pt, pt, BlockSize * ThreadSize * 128 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_mk, mk, BlockSize * ThreadSize * 16 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_tag, tag, BlockSize * ThreadSize * 16 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_aad, aad, BlockSize * ThreadSize * 112 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_iv, iv, BlockSize * ThreadSize * 12 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy((void**)g_ct, ct, BlockSize * ThreadSize * 128 * sizeof(uint8_t), hipMemcpyHostToDevice);
    //hipMemcpy((void**)g_ctx, ctx, BlockSize* ThreadSize * sizeof(LEA_GCM_CTX), hipMemcpyHostToDevice);


    //성능측정
    hipEvent_t start, stop;
    float elapsed_time_ms = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Global
    GCM_8bit_table << < BlockSize, ThreadSize >> > (g_mk, mk_len, g_iv, iv_len, g_aad, aad_len, g_ct, g_pt, pt_len, g_tag, tag_len);

    hipDeviceSynchronize();
    //성능측정
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    printf("8bit table version code\n");
    printf("elapsed_time_ms is %4.4f\n", elapsed_time_ms);
    printf("Performance : %4.2f GCM time per second \n", BlockSize * ThreadSize / ((elapsed_time_ms / 1000)));

    //cuda->cpu
    hipMemcpy(tag, g_tag, BlockSize * 16 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(aad, g_aad, BlockSize * 112 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(ct, g_ct, BlockSize * 128 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(pt, g_pt, BlockSize * 128 * ThreadSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    //hipMemcpy(ctx, g_ctx, BlockSize  * ThreadSize * sizeof(LEA_GCM_CTX), hipMemcpyDeviceToHost);



    hipFree(g_pt);
    hipFree(g_tag);
    hipFree(g_aad);
    hipFree(g_mk);
    hipFree(g_ct);
    //hipFree(g_ctx);
    hipFree(g_iv);

    free(pt);
    //free(ctx);
    free(aad);
    free(mk);
    free(iv);
    free(tag);
    free(ct);
}

int main() {
    test_GPU_parallel(64, 32);
    //printf("\n\n");
    test_GPU_REF(16, 32);
    //printf("\n\n");
    test_GPU_4bit_table(16, 32);
    //printf("\n\n");
    test_GPU_8bit_table(16, 32);
}


